#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#define BLOCK_SIZE 512
#define EL_PER_THREAD 1

extern "C"
{


    texture<unsigned char, 2> tex;

//    texture<float, 2, hipReadModeElementType> texData;

    typedef struct frag {
        int* pos;
        int* sub_pos; // position ( sub frags)
        int* id_c;
        int* start_bp;
        int* len_bp;
        int* sub_len; // length(sub frags)
        int* circ;
        int* id;
        int* prev;
        int* next;
        int* l_cont;
        int* sub_l_cont; // length(sub frags)
        int* l_cont_bp;
        int* ori;
        int* rep;
        int* activ;
        int* id_d;
    } frag;



//    typedef struct double14{
//        double x0;
//        double x1;
//        double x2;
//        double x3;
//        double x4;
//        double x5;
//        double x6;
//        double x7;
//        double x8;
//        double x9;
//        double x10;
//        double x11;
//        double x12;
//        double x13;
//        double x14;
//    } double14;

//    typedef struct bigfrag {
//        int* pos;
//        int* id_c;
//        int* start_bp;
//        int* circ;
//        int* l_cont_bp;
//        int* ori;
//        int* rep;
//        int* activ;
//    } bigfrag;

    typedef struct __attribute__ ((packed)) param_simu {
        float kuhn __attribute__ ((packed));
        float lm __attribute__ ((packed));
        float c1 __attribute__ ((packed));
        float slope __attribute__ ((packed));
        float d __attribute__ ((packed));
        float d_max __attribute__ ((packed));
        float fact __attribute__ ((packed));
        float v_inter __attribute__ ((packed));
    } param_simu;


    __device__ double atomicAdd(double* address, double val)
    {
        unsigned long long int* address_as_ull = (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;
        do {
            assumed = old;
            old = atomicCAS(address_as_ull, assumed,
                            __double_as_longlong(val +
                                                 __longlong_as_double(assumed)));
        } while (assumed != old);
        return __longlong_as_double(old);
    }
    __global__ void init_rng(int nthreads, hiprandState *s, unsigned long long seed, unsigned long long offset)
    {
        int id = blockIdx.x*blockDim.x + threadIdx.x;

        if (id >= nthreads)
            return;
        hiprand_init(seed, id, offset, &s[id]);
    }

    __device__ float factorial(float n)
    {

        float result = 1;
        n = floor(n);
        if (n<10) {
            for(int c = 1 ; c <= n ; c++ )
                result = result * c;
        }
        else {
            result = powf(n,n) * exp(-n) * sqrtf(2 * M_PI * n);
        }
        return ( result );
    }

    __device__ param_simu modify_param_simu(int id_modifier, param_simu p, float var)
    {
        param_simu out;
        if (id_modifier == 0) {
            out.kuhn = p.kuhn;
            out.lm = p.lm;
            out.slope = p.slope;
            out.d = var;
            out.c1 = p.c1;
            out.d_max = p.d_max;
            out.v_inter = p.v_inter;
            out.fact = p.fact;
        }
        else if (id_modifier == 1) {
            out.kuhn = p.kuhn;
            out.lm = p.lm;
            out.slope = var;
            out.d = p.d;
            out.c1 = p.c1;
            out.d_max = p.d_max;
            out.v_inter = p.v_inter;
            out.fact = p.fact;
        }
        return (out);
    }


    __device__ float rippe_contacts(float s, param_simu p)
    {
        // s = distance in kb
        // p = model's parameters
        float result = 0.0f;
        if ((s>0.0f) && (s<p.d_max)) {
            result = (p.c1 * pow(s, p.slope) * exp((p.d-2)/(pow(s*p.lm/p.kuhn, 2.0f ) + p.d)  )) * p.fact;
        }
        float out = max(result, p.v_inter);
        return ( out );
    }

//    __device__ float rippe_contacts_circ(float s, float s_tot, param_simu p)
//    {
//        // s = distance in kb
//        // p = model's parameters
//        // s_tot = total length of circular contig
//        float result = 0.0f;
//        float n_dist = 1.0f;
//        float n_tot = 1.0f;
//        float n = 1.0f;
//        float K = 1.0f;
//        float norm_circ, norm_lin, nmax, val;
//        if ((s > 0.0f) && (s < p.d_max)){
////        if ((s < s_tot) && (s > 0.0) && (s < p.d_max)){
//            K = p.lm / p.kuhn;
//            n_dist = s ;
//            n_tot = s_tot;
//            nmax = K * 1;
//
//            n = K * n_dist *(n_tot - n_dist) / n_tot;
//
//            norm_lin = rippe_contacts(s, p);
//            norm_circ = (powf(p.kuhn, -3.0f) * powf(nmax, p.slope) * expf((p.d - 2.0f)/(powf(nmax, 2.0f ) + p.d))) * p.fact;
//
//            val = (powf(p.kuhn, -3.0f) * powf(n, p.slope) * expf((p.d - 2.0f)/(powf(n, 2.0f ) + p.d))) * p.fact;
//            result = val * norm_lin / norm_circ;
//            result = val;
//        }
//        float out = max(result, p.v_inter);
////        else{
////            result = p.v_inter;
////        }
//        return ( out );
//    }


    __device__ float rippe_contacts_circ(float s, float s_tot, param_simu p)
    {
        // s = distance in kb
        // p = model's parameters
        // s_tot = total length of circular contig
        float result = 0.0f;
        float n_dist = 1.0f;
        float n_tot = 1.0f;
        float n = 1.0f;
        float K = 1.0f;
        if ((s > 0.0f) && (s < p.d_max)) {
//        if ((s < s_tot) && (s > 0.0) && (s < p.d_max)){
            K = p.lm / p.kuhn;
            n_dist = s ;
            n_tot = s_tot;
            n = K * n_dist *(n_tot - n_dist) / n_tot;

            result = (powf(p.kuhn, -3.0f) * powf(n, p.slope) * expf((p.d - 2.0f)/(powf(n, 2.0f ) + p.d))) * p.fact;
        }
        float out = max(result, p.d_max);
//        else{
//            result = p.v_inter;
//        }
        return ( out );

    }


    __device__ float evaluate_likelihood_pxl_float(float ex, float ob)
    {
        // ex = expected n contacts
        // ob = observed n contacts
        float res = 0.0;
        float lim = 15.0;
        if (ex != 0.0) {
            if (ob >=lim) {
                res = ob * log10(ex) - ex - (ob * log10(ob) - ob + log10(sqrtf(ob * 2.0f * M_PI)));
            }
            else if ((ob>0) && (ob<lim)) {
                res = ob * log10(ex) - ex - log10(factorial(ob));
            }
            else if (ob==0) {
                res = - ex;
            }
        }

        return (res);
    }



    __device__ double evaluate_likelihood_pxl_double(double ex, double ob)
    {
        // ex = expected n contacts
        // ob = observed n contacts
        double res = 0;
        double lim = 15;
        if (ex != 0) {
            if (ob >=lim) {
                res = ob * log10(ex) - ex - (ob * log10(ob) - ob + log10(sqrt(ob * 2.0 * M_PI)));
            }
            else if ((ob>0) && (ob<lim)) {
                res = ob * log10(ex) - ex - log10((double) factorial((float) ob));
            }
            else if (ob==0) {
                res = - ex;
            }
        }

        return (res);
    }


    __device__ int2 lin_2_2dpos(int ind)
    {
        int i = ind + 1;
        int x = (-0.5 + 0.5 * sqrt((float) 1 + 8 * (i - 1))) + 2;
        int y =  x * (3 - x) / 2 + i - 1;
        //int2 out = (int2) (x - 1,y - 1);
        int2 out;
        out.x = min(x - 1, y -1);
        out.y = max(x - 1, y - 1);
        return (out);
    }

    __device__ int conv_plan_pos_2_lin(int2 pos)
    {
        int x = pos.x + 1;
        int y = pos.y + 1;
        int i = min(x,y);
        int j = max(x,y);
        int ind = (j * (j - 3)) / 2 + i;
//        int ind = (j * (j - 3) / 2 + i + 1) - 1;
        return ind;
    }


//        __global__void big_sum_14(float14 *out, float14 * input,unsigned int seq_count, unsigned int n)
//        {
//          // Needs to be variable-size to prevent the braindead CUDA compiler from
//          // running constructors on this array. Grrrr.
//          extern __shared__ double14 sdata[];
//
//          unsigned int tid = threadIdx.x;
//
//          unsigned int i = blockIdx.x*BLOCK_SIZE*seq_count + tid;
//
//          double14 acc = {0.0};
//          for (unsigned s = 0; s < seq_count; ++s)
//          {
//            if (i >= n)
//              break;
//            acc = REDUCE(acc, READ_AND_MAP(i));
//
//            i += BLOCK_SIZE;
//          }
//
//          sdata[tid] = acc;
//
//          __syncthreads();
//
//          #if (BLOCK_SIZE >= 512)
//            if (tid < 256) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 256]); }
//            __syncthreads();
//          #endif
//
//          #if (BLOCK_SIZE >= 256)
//            if (tid < 128) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 128]); }
//            __syncthreads();
//          #endif
//
//          #if (BLOCK_SIZE >= 128)
//            if (tid < 64) { sdata[tid] = REDUCE(sdata[tid], sdata[tid + 64]); }
//            __syncthreads();
//          #endif
//
//          if (tid < 32)
//          {
//            // 'volatile' required according to Fermi compatibility guide 1.2.2
//            volatile out_type *smem = sdata;
//            if (BLOCK_SIZE >= 64) smem[tid] = REDUCE(smem[tid], smem[tid + 32]);
//            if (BLOCK_SIZE >= 32) smem[tid] = REDUCE(smem[tid], smem[tid + 16]);
//            if (BLOCK_SIZE >= 16) smem[tid] = REDUCE(smem[tid], smem[tid + 8]);
//            if (BLOCK_SIZE >= 8)  smem[tid] = REDUCE(smem[tid], smem[tid + 4]);
//            if (BLOCK_SIZE >= 4)  smem[tid] = REDUCE(smem[tid], smem[tid + 2]);
//            if (BLOCK_SIZE >= 2)  smem[tid] = REDUCE(smem[tid], smem[tid + 1]);
//          }
//
//          if (tid == 0) out[blockIdx.x] = sdata[0];
//        }



    __global__ void select_them(const int* __restrict__ spData_row,
                                const int* __restrict__ spData_col,
                                const int* __restrict__ vect_id_c,
                                int *selec,
                                int id_ctg1,
                                int id_ctg2,
                                int *counter,
                                int size_arr)
    {
        __shared__ int selec_smem[512];
        __shared__ int counter_smem;
        int *counter_smem_ptr;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int condition = idx < size_arr;
        int curr_id_ctg1, curr_id_ctg2, fi, fj;
        int local_count = 0;
        if ((threadIdx.x == 0) && (condition ==1))
        {
            counter_smem_ptr = &counter_smem;
            counter_smem = 0;
        }

        selec_smem[threadIdx.x] = -1;

        __syncthreads();

        // each counting thread writes its index to shared memory

        if (condition == 1) {
            fi = spData_row[idx];
            fj = spData_col[idx];
            curr_id_ctg1 = vect_id_c[fi];
            // each counting thread writes its index to shared memory //
            if ((curr_id_ctg1 == id_ctg1) || (curr_id_ctg1 == id_ctg2)) {
                curr_id_ctg2 = vect_id_c[fj];
                if ((curr_id_ctg2 == id_ctg1) || (curr_id_ctg2 == id_ctg2)) {
                    local_count = atomicAdd(counter_smem_ptr, 1);
                    selec_smem[local_count] =  idx;
                }
            }
        }

        __syncthreads();

        if (threadIdx.x == 0)
            counter_smem = atomicAdd(counter, counter_smem);

        __syncthreads();


        if ((selec_smem[threadIdx.x] >= 0) && (condition ==1))
            selec[counter_smem + threadIdx.x] = selec_smem[threadIdx.x];

    }




    __global__ void select_uniq_id_c(frag* fragArray,
                                     int* list_uniq_id_c,
                                     int* list_uniq_len,
                                     int* counter,
                                     int n_frags)
    {
        __shared__ int selec_smem[1024]; //2 * 512
        __shared__ int counter_smem;
        int *counter_smem_ptr;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int condition = idx < n_frags;
        int local_count = 0;
        int pos, len, id_c;
        if ((threadIdx.x == 0) && (condition ==1))
        {
            counter_smem_ptr = &counter_smem;
            counter_smem = 0;
        }

        selec_smem[threadIdx.x * 2] = -1;

        __syncthreads();

        // each counting thread writes its index to shared memory

        if (condition == 1) {
            pos = fragArray->pos[idx];
            // each counting thread writes its index to shared memory //
            if (pos == 0) {
                len = fragArray->l_cont[idx];
                id_c = fragArray->id_c[idx];
                local_count = atomicAdd(counter_smem_ptr, 1);
                selec_smem[local_count * 2] =  id_c;
                selec_smem[local_count * 2 + 1] =  len;
            }
        }

        __syncthreads();

        if (threadIdx.x == 0)
            counter_smem = atomicAdd(counter, counter_smem);

        __syncthreads();


        if ((selec_smem[threadIdx.x * 2] >= 0) && (condition ==1)) {
            list_uniq_id_c[counter_smem + threadIdx.x] = selec_smem[threadIdx.x * 2];
            list_uniq_len[counter_smem + threadIdx.x] = selec_smem[threadIdx.x * 2 + 1];
        }
    }


    __global__ void explode_genome(frag* fragArray,
                                   int n_frags)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int condition = idx < n_frags;
        if (condition) {
            fragArray->pos[idx] = 0;
            fragArray->start_bp[idx] = 0;
            fragArray->sub_pos[idx] = 0;
            fragArray->id_c[idx] = idx;
            fragArray->prev[idx] = -1;
            fragArray->next[idx] = -1;
            fragArray->l_cont[idx] = 1;
            fragArray->l_cont_bp[idx] = fragArray->len_bp[idx];
            fragArray->sub_l_cont[idx] = fragArray->sub_len[idx];
        }
    }


    __global__ void count_num(int* list_vals,
                              int value,
                              int* counter,
                              int n_values)
    {
        __shared__ int selec_smem[1024]; //2 * 512
        __shared__ int counter_smem;
        int *counter_smem_ptr;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int condition = idx < n_values;
        int local_count = 0;
        int val;;
        if ((threadIdx.x == 0) && (condition ==1))
        {
            counter_smem_ptr = &counter_smem;
            counter_smem = 0;
        }

        selec_smem[threadIdx.x * 2] = -1;

        __syncthreads();

        // each counting thread writes its index to shared memory

        if (condition == 1) {
            val = list_vals[idx];
            // each counting thread writes its index to shared memory //
            if (val == value) {
                local_count = atomicAdd(counter_smem_ptr, 1);
            }
        }
        __syncthreads();
        if (threadIdx.x == 0)
            counter_smem = atomicAdd(counter, counter_smem);
    }



    __global__ void make_old_2_new_id_c(int* list_uniq_id_c,
                                        int* list_old_2_new_id_c,
                                        int* counter)
    {

        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int condition = idx < counter[0];
        int id_c;
        if (condition ==1) {
            id_c = list_uniq_id_c[idx];
            list_old_2_new_id_c[id_c] = idx;
        }
    }


    __global__ void slice_sp_mat(const int* __restrict__ spData_dat,
                                 const int* __restrict__ spData_row,
                                 const int* __restrict__ spData_col,
                                 const int* __restrict__ vect_id_c,
                                 int* sub_spData_row,
                                 int* sub_spData_col,
                                 int* sub_spData_dat,
                                 int id_ctg1,
                                 int id_ctg2,
                                 int *counter,
                                 int size_arr)
    {

//        __shared__ int selec_smem[1536]; //512 * 3
//        __shared__ int selec_smem[768]; // 256 * 3
        __shared__ int selec_smem[384]; // 128 * 3
        __shared__ int counter_smem;

        int *counter_smem_ptr;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int curr_id_ctg1, curr_id_ctg2, fi, fj, dat;
        int tid3 = threadIdx.x * 3;
        int local_count = 0;
        int condition = idx < size_arr;

        if ((threadIdx.x == 0))
        {
            counter_smem_ptr = &counter_smem;
            counter_smem = 0;
        }

        selec_smem[tid3] = -1;

        __syncthreads();

        if (condition == 1) {
            fi = spData_row[idx];
            fj = spData_col[idx];
            dat = spData_dat[idx];
            curr_id_ctg1 = vect_id_c[fi];
            // each counting thread writes its index to shared memory //
            if ((curr_id_ctg1 == id_ctg1) || (curr_id_ctg1 == id_ctg2)) {
                curr_id_ctg2 = vect_id_c[fj];
                if ((curr_id_ctg2 == id_ctg1) || (curr_id_ctg2 == id_ctg2)) {
                    local_count = atomicAdd(counter_smem_ptr, 1);
                    selec_smem[local_count * 3] =  dat;
                    selec_smem[local_count * 3 + 1] = fi;
                    selec_smem[local_count * 3 + 2] = fj;
                }
            }
        }

        __syncthreads();

        if ((threadIdx.x == 0) && (condition == 1))
            counter_smem = atomicAdd(counter, counter_smem);
        __syncthreads();

        if (selec_smem[tid3] > 0) {
            sub_spData_dat[counter_smem + threadIdx.x] = selec_smem[tid3];
            sub_spData_row[counter_smem + threadIdx.x] = selec_smem[tid3 + 1];
            sub_spData_col[counter_smem + threadIdx.x] = selec_smem[tid3 + 2];
        }
    }






    __global__ void flip_frag(frag* fragArray,frag* o_fragArray, int id_f_flip,
                              int n_frags, float2 subfrags)
    {
        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (id_frag < n_frags) {

            int contig_fi = o_fragArray->id_c[id_frag];
            int pos_fi = o_fragArray->pos[id_frag];
            // UDPATE
            int sub_pos_fi = o_fragArray->sub_pos[id_frag];
            // UDPATE
            int l_cont_fi = o_fragArray->l_cont[id_frag];
            // UDPATE
            int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag];
            // UDPATE
            int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
            int len_bp_fi = o_fragArray->len_bp[id_frag];
            // UDPATE
            int sub_len_fi = o_fragArray->sub_len[id_frag];
            // UDPATE
            int circ_fi = o_fragArray->circ[id_frag];
            int id_prev_fi = o_fragArray->prev[id_frag];
            int id_next_fi = o_fragArray->next[id_frag];
            int start_bp_fi = o_fragArray->start_bp[id_frag];
            int or_fi = o_fragArray->ori[id_frag];
            int rep_fi = o_fragArray->rep[id_frag];
            int activ_fi = o_fragArray->activ[id_frag];
            int id_d_fi = o_fragArray->id_d[id_frag];

            fragArray->pos[id_frag] = pos_fi;

            fragArray->sub_pos[id_frag] = sub_pos_fi;

            fragArray->id_c[id_frag] = contig_fi;
            fragArray->start_bp[id_frag] = start_bp_fi;
            fragArray->len_bp[id_frag] = len_bp_fi;
            // UDPATE
            fragArray->sub_len[id_frag] = sub_len_fi;
            // UDPATE
            fragArray->circ[id_frag] = circ_fi;
            fragArray->id[id_frag] = id_frag;
            if (id_frag == id_f_flip) {
                fragArray->ori[id_frag] = or_fi * -1;
            }
            else {
                fragArray->ori[id_frag] = or_fi;
            }
            fragArray->prev[id_frag] = id_prev_fi;
            fragArray->next[id_frag] = id_next_fi;
            fragArray->l_cont[id_frag] = l_cont_fi;
            // UDPATE
            fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;
            // UDPATE
            fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
            fragArray->rep[id_frag] = rep_fi;
            fragArray->activ[id_frag] = activ_fi;
            fragArray->id_d[id_frag] = id_d_fi;
        }
    }



    __global__ void swap_activity_frag(frag* fragArray,frag* o_fragArray, int id_f_unactiv, int max_id_contig,
                                       int n_frags)
    {
        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (id_frag < n_frags) {

            int contig_fi = o_fragArray->id_c[id_frag];
            int pos_fi = o_fragArray->pos[id_frag];
            // UDPATE
            int sub_pos_fi = o_fragArray->sub_pos[id_frag];
            // UDPATE
            int l_cont_fi = o_fragArray->l_cont[id_frag];
            // UDPATE
            int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag];
            // UDPATE
            int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
            int len_bp_fi = o_fragArray->len_bp[id_frag];
            // UDPATE
            int sub_len_fi = o_fragArray->sub_len[id_frag];
            // UDPATE
            int circ_fi = o_fragArray->circ[id_frag];
            int id_prev_fi = o_fragArray->prev[id_frag];
            int id_next_fi = o_fragArray->next[id_frag];
            int start_bp_fi = o_fragArray->start_bp[id_frag];
            int or_fi = o_fragArray->ori[id_frag];
            int rep_fi = o_fragArray->rep[id_frag];
            int activ_fi = o_fragArray->activ[id_frag];
            int id_d_fi = o_fragArray->id_d[id_frag];

            fragArray->pos[id_frag] = pos_fi;

            fragArray->sub_pos[id_frag] = sub_pos_fi;

            fragArray->start_bp[id_frag] = start_bp_fi;
            fragArray->len_bp[id_frag] = len_bp_fi;
            fragArray->circ[id_frag] = circ_fi;
            fragArray->id[id_frag] = id_frag;
            fragArray->ori[id_frag] = or_fi;
//            if ((id_frag == id_f_unactiv) && (id_d_fi != id_frag)){
            if ((id_frag == id_f_unactiv) && (rep_fi == 1)) {
                fragArray->activ[id_frag] = 0 * (activ_fi == 1) + 1 * (activ_fi == 0);
                fragArray->id_c[id_frag] = contig_fi * (activ_fi == 1) + (max_id_contig + 1) * (activ_fi == 0);
//                fragArray->id_c[id_frag] = contig_fi * (activ_fi == 1) + (max_id_contig + 1) * (activ_fi == 0);
            }
            else {
                fragArray->activ[id_frag] = activ_fi;
                fragArray->id_c[id_frag] = contig_fi;
            }
            fragArray->prev[id_frag] = id_prev_fi;
            fragArray->next[id_frag] = id_next_fi;
            fragArray->l_cont[id_frag] = l_cont_fi;
            // UDPATE
            fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;
            // UDPATE
            fragArray->sub_len[id_frag] = sub_len_fi;
            // UDPATE
            fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
            fragArray->rep[id_frag] = rep_fi;
            fragArray->id_d[id_frag] = id_d_fi;
        }
    }


    __global__ void pop_out_frag(frag* fragArray,frag* o_fragArray, int* pop_id_contigs, int id_f_pop,
                                 int max_id_contig, int n_frags)
    {
        __shared__ int contig_f_pop;
        __shared__ int pos_f_pop;
        // UDPATE
        __shared__ int sub_pos_f_pop;
        // UDPATE
        __shared__ int l_cont_f_pop;
        // UDPATE
        __shared__ int sub_l_cont_f_pop;
        // UDPATE
        __shared__ int l_cont_bp_f_pop;
        __shared__ int len_bp_f_pop;
        // UDPATE
        __shared__ int sub_len_f_pop;
        // UDPATE
        __shared__ int start_bp_f_pop;
        __shared__ int id_prev_f_pop;
        __shared__ int id_next_f_pop;
        __shared__ int or_f_pop;
        __shared__ int circ_f_pop;


        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            contig_f_pop = o_fragArray->id_c[id_f_pop];
            pos_f_pop = o_fragArray->pos[id_f_pop];
            // UDPATE
            sub_pos_f_pop = o_fragArray->sub_pos[id_f_pop];
            // UDPATE
            l_cont_f_pop = o_fragArray->l_cont[id_f_pop];
            // UDPATE
            sub_l_cont_f_pop = o_fragArray->sub_l_cont[id_f_pop];
            // UDPATE
            l_cont_bp_f_pop = o_fragArray->l_cont_bp[id_f_pop];
            len_bp_f_pop = o_fragArray->len_bp[id_f_pop];
            // UDPATE
            sub_len_f_pop = o_fragArray->sub_len[id_f_pop];
            // UDPATE
            start_bp_f_pop = o_fragArray->start_bp[id_f_pop];
            id_prev_f_pop = o_fragArray->prev[id_f_pop];
            id_next_f_pop = o_fragArray->next[id_f_pop];
            or_f_pop = o_fragArray->ori[id_f_pop];
            circ_f_pop = o_fragArray->circ[id_f_pop];
        }
        __syncthreads();

        if (id_frag < n_frags) {
            int contig_fi = o_fragArray->id_c[id_frag];
            int pos_fi = o_fragArray->pos[id_frag];
            // UDPATE
            int sub_pos_fi = o_fragArray->sub_pos[id_frag];
            // UDPATE
            int l_cont_fi = o_fragArray->l_cont[id_frag];
            // UDPATE
            int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag];
            // UDPATE
            int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
            int len_bp_fi = o_fragArray->len_bp[id_frag];
            // UDPATE
            int sub_len_fi = o_fragArray->sub_len[id_frag];
            // UDPATE
            int circ_fi = o_fragArray->circ[id_frag];
            int id_prev_fi = o_fragArray->prev[id_frag];
            int id_next_fi = o_fragArray->next[id_frag];
            int start_bp_fi = o_fragArray->start_bp[id_frag];
            int or_fi = o_fragArray->ori[id_frag];
            int rep_fi = o_fragArray->rep[id_frag];
            int id_d_fi = o_fragArray->id_d[id_frag];
            int activ_fi = o_fragArray->activ[id_frag];
            if (l_cont_f_pop > 2) {
                if ( contig_fi == contig_f_pop) {
                    if (pos_fi < pos_f_pop) {
                        fragArray->pos[id_frag] = pos_fi;
                        // UDPATE
                        fragArray->sub_pos[id_frag] = sub_pos_fi;
                        // UDPATE
                        fragArray->id_c[id_frag] = contig_fi;
                        pop_id_contigs[id_frag] = contig_fi;
                        fragArray->start_bp[id_frag] = start_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;

                        fragArray->sub_len[id_frag] = sub_len_fi;

                        fragArray->circ[id_frag] = circ_fi;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
//                        fragArray->prev[id_frag] = id_prev_fi;
                        if ((id_frag == id_next_f_pop) && (circ_f_pop == 1)) {
                            fragArray->prev[id_frag] = id_prev_f_pop;
                        }
                        else {
                            fragArray->prev[id_frag] = id_prev_fi;
                        }
                        if (pos_fi == (pos_f_pop - 1)) {
                            fragArray->next[id_frag] = id_next_f_pop;
                        }
                        else {
                            fragArray->next[id_frag] = id_next_fi;
                        }
                        fragArray->l_cont[id_frag] = l_cont_fi -1;

                        fragArray->sub_l_cont[id_frag] = sub_l_cont_fi - sub_len_f_pop;

                        fragArray->l_cont_bp[id_frag] = l_cont_bp_fi - len_bp_f_pop;
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;
                    }
                    else if (pos_fi == pos_f_pop) {
                        fragArray->pos[id_frag] = 0;
                        // UDPATE
                        fragArray->sub_pos[id_frag] = 0;
                        // UDPATE
                        fragArray->id_c[id_frag] = max_id_contig + 1;
                        pop_id_contigs[id_frag] = max_id_contig + 1;
                        fragArray->start_bp[id_frag] = 0;
                        fragArray->len_bp[id_frag] = len_bp_fi;

                        fragArray->sub_len[id_frag] = sub_len_fi;

                        fragArray->circ[id_frag] = 0;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = 1;
                        fragArray->prev[id_frag] = -1;
                        fragArray->next[id_frag] = -1;
                        fragArray->l_cont[id_frag] = 1;

                        fragArray->sub_l_cont[id_frag] = sub_len_fi;

                        fragArray->l_cont_bp[id_frag] = len_bp_fi;
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;

                    }
                    else if (pos_fi > pos_f_pop) {
                        fragArray->pos[id_frag] = pos_fi - 1;
                        // UDPATE
                        fragArray->sub_pos[id_frag] = sub_pos_fi - sub_len_f_pop;
                        // UDPATE
                        fragArray->id_c[id_frag] = contig_fi;
                        pop_id_contigs[id_frag] = contig_fi;
                        fragArray->start_bp[id_frag] = start_bp_fi - len_bp_f_pop;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        // UDPATE
                        fragArray->sub_len[id_frag] = sub_len_fi;
                        // UDPATE
                        fragArray->circ[id_frag] = circ_fi;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        if (pos_fi == (pos_f_pop + 1)) {
                            fragArray->prev[id_frag] = id_prev_f_pop;
                        }
                        else {
                            fragArray->prev[id_frag] = id_prev_fi;
                        }
                        if ((id_frag == id_prev_f_pop) && (circ_f_pop == 1)) {
                            fragArray->next[id_frag] = id_next_f_pop;
                        }
                        else {
                            fragArray->next[id_frag] = id_next_fi;
                        }
//                        fragArray->next[id_frag] = id_next_fi;
                        fragArray->l_cont[id_frag] = l_cont_fi -1 ;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_fi - sub_len_f_pop;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_fi - len_bp_f_pop;
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;
                    }

                }
                else {
                    fragArray->pos[id_frag] = pos_fi;
                    // UDPATE
                    fragArray->sub_pos[id_frag] = sub_pos_fi;
                    // UDPATE
                    fragArray->id_c[id_frag] = contig_fi;
                    pop_id_contigs[id_frag] = contig_fi;
                    fragArray->start_bp[id_frag] = start_bp_fi;
                    fragArray->len_bp[id_frag] = len_bp_fi;
                    // UDPATE
                    fragArray->sub_len[id_frag] = sub_len_fi;
                    // UDPATE
                    fragArray->circ[id_frag] = circ_fi;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = or_fi;
                    fragArray->prev[id_frag] = id_prev_fi;
                    fragArray->next[id_frag] = id_next_fi;
                    fragArray->l_cont[id_frag] = l_cont_fi;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                    // UDPATE
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;
                    // UDPATE
                    fragArray->rep[id_frag] = rep_fi;
                    fragArray->activ[id_frag] = activ_fi;
                    fragArray->id_d[id_frag] = id_d_fi;

                }
            }
            else if (l_cont_f_pop == 2) {
                if ( contig_fi == contig_f_pop) {
                    if (pos_fi < pos_f_pop) {
                        fragArray->pos[id_frag] = pos_fi;
                        // UDPATE
                        fragArray->sub_pos[id_frag] = sub_pos_fi;
                        // UDPATE
                        fragArray->id_c[id_frag] = contig_fi;
                        pop_id_contigs[id_frag] = contig_fi;
                        fragArray->start_bp[id_frag] = start_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        // UDPATE
                        fragArray->sub_len[id_frag] = sub_len_fi;
                        // UDPATE
                        fragArray->circ[id_frag] = 0;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        fragArray->prev[id_frag] = -1;
                        fragArray->next[id_frag] = -1;
                        fragArray->l_cont[id_frag] = l_cont_fi -1;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_fi - len_bp_f_pop;
                        // UDPATE
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_fi - sub_len_f_pop;
                        // UDPATE
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;
                    }
                    else if (pos_fi == pos_f_pop) {
                        fragArray->pos[id_frag] = 0;
                        // UDPATE
                        fragArray->sub_pos[id_frag] = 0;
                        // UDPATE
                        fragArray->id_c[id_frag] = max_id_contig + 1;
                        pop_id_contigs[id_frag] = max_id_contig + 1;
                        fragArray->start_bp[id_frag] = 0;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        // UDPATE
                        fragArray->sub_len[id_frag] = sub_len_fi;
                        // UDPATE
                        fragArray->circ[id_frag] = 0;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = 1;
                        fragArray->prev[id_frag] = -1;
                        fragArray->next[id_frag] = -1;
                        fragArray->l_cont[id_frag] = 1;
                        fragArray->l_cont_bp[id_frag] = len_bp_fi;
                        // UDPATE
                        fragArray->sub_l_cont[id_frag] = sub_len_fi;
                        // UDPATE
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;

                    }
                    else if (pos_fi > pos_f_pop) {
                        fragArray->pos[id_frag] = pos_fi - 1;
                        // UDPATE
                        fragArray->sub_pos[id_frag] = sub_pos_fi - sub_len_f_pop;
                        // UDPATE
                        fragArray->id_c[id_frag] = contig_fi;
                        pop_id_contigs[id_frag] = contig_fi;
                        fragArray->start_bp[id_frag] = start_bp_fi - len_bp_f_pop;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        // UDPATE
                        fragArray->sub_len[id_frag] = sub_len_fi;
                        // UDPATE
                        fragArray->circ[id_frag] = 0;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        fragArray->prev[id_frag] = -1;
                        fragArray->next[id_frag] = -1;
                        fragArray->l_cont[id_frag] = l_cont_fi -1 ;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_fi - len_bp_f_pop;
                        // UDPATE
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_fi - sub_len_f_pop;
                        // UDPATE
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;
                    }

                }
                else {
                    fragArray->pos[id_frag] = pos_fi;
                    // UDPATE
                    fragArray->sub_pos[id_frag] = sub_pos_fi;
                    // UDPATE
                    fragArray->id_c[id_frag] = contig_fi;
                    pop_id_contigs[id_frag] = contig_fi;
                    fragArray->start_bp[id_frag] = start_bp_fi;
                    fragArray->len_bp[id_frag] = len_bp_fi;
                    // UDPATE
                    fragArray->sub_len[id_frag] = sub_len_fi;
                    // UDPATE
                    fragArray->circ[id_frag] = circ_fi;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = or_fi;
                    fragArray->prev[id_frag] = id_prev_fi;
                    fragArray->next[id_frag] = id_next_fi;
                    fragArray->l_cont[id_frag] = l_cont_fi;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                    // UDPATE
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;
                    // UDPATE
                    fragArray->rep[id_frag] = rep_fi;
                    fragArray->activ[id_frag] = activ_fi;
                    fragArray->id_d[id_frag] = id_d_fi;

                }
            }
            else {
                fragArray->pos[id_frag] = pos_fi;
                // UDPATE
                fragArray->sub_pos[id_frag] = sub_pos_fi;
                // UDPATE
                fragArray->id_c[id_frag] = contig_fi;
                pop_id_contigs[id_frag] = contig_fi;
                fragArray->start_bp[id_frag] = start_bp_fi;
                fragArray->len_bp[id_frag] = len_bp_fi;
                // UDPATE
                fragArray->sub_len[id_frag] = sub_len_fi;
                // UDPATE
                fragArray->circ[id_frag] = circ_fi;
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = or_fi;
                fragArray->prev[id_frag] = id_prev_fi;
                fragArray->next[id_frag] = id_next_fi;
                fragArray->l_cont[id_frag] = l_cont_fi;
                // UDPATE
                fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;
                // UDPATE
                fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                fragArray->rep[id_frag] = rep_fi;
                fragArray->activ[id_frag] = activ_fi;
                fragArray->id_d[id_frag] = id_d_fi;
            }
        }
    }


    __global__ void pop_in_frag_1(frag* fragArray,frag* o_fragArray, int id_f_pop, int id_f_ins, int max_id_contig,
                                  int ori_f_pop,
                                  int n_frags)
    // split insert @ left
    {
        __shared__ int contig_f_pop;
        __shared__ int pos_f_pop;
        __shared__ int sub_pos_f_pop;             // UDPATE
        __shared__ int l_cont_f_pop;
        __shared__ int sub_l_cont_f_pop;            // UDPATE
        __shared__ int l_cont_bp_f_pop;
        __shared__ int len_bp_f_pop;
        __shared__ int sub_len_f_pop;             // UDPATE
        __shared__ int start_bp_f_pop;
        __shared__ int id_prev_f_pop;
        __shared__ int id_next_f_pop;
        __shared__ int activ_f_pop;
//        __shared__ int or_f_pop;

        __shared__ int contig_f_ins;
        __shared__ int pos_f_ins;
        __shared__ int sub_pos_f_ins;              // UDPATE
        __shared__ int l_cont_f_ins;
        __shared__ int sub_l_cont_f_ins;              // UDPATE
        __shared__ int l_cont_bp_f_ins;
        __shared__ int len_bp_f_ins;
        __shared__ int sub_len_f_ins;              // UDPATE
        __shared__ int start_bp_f_ins;
        __shared__ int id_prev_f_ins;
        __shared__ int id_next_f_ins;
        __shared__ int circ_f_ins;
        __shared__ int or_f_ins;
        __shared__ int activ_f_ins;
        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            contig_f_pop = o_fragArray->id_c[id_f_pop];
            pos_f_pop = o_fragArray->pos[id_f_pop];
            sub_pos_f_pop = o_fragArray->sub_pos[id_f_pop];              // UDPATE
            l_cont_f_pop = o_fragArray->l_cont[id_f_pop];
            sub_l_cont_f_pop = o_fragArray->sub_l_cont[id_f_pop];              // UDPATE
            l_cont_bp_f_pop = o_fragArray->l_cont_bp[id_f_pop];
            len_bp_f_pop = o_fragArray->len_bp[id_f_pop];
            sub_len_f_pop = o_fragArray->sub_len[id_f_pop];              // UDPATE
            start_bp_f_pop = o_fragArray->start_bp[id_f_pop];
            id_prev_f_pop = o_fragArray->prev[id_f_pop];
            id_next_f_pop = o_fragArray->next[id_f_pop];
            activ_f_pop = o_fragArray->activ[id_f_pop];

            contig_f_ins = o_fragArray->id_c[id_f_ins];
            pos_f_ins = o_fragArray->pos[id_f_ins];
            sub_pos_f_ins = o_fragArray->sub_pos[id_f_ins];              // UDPATE
            l_cont_f_ins = o_fragArray->l_cont[id_f_ins];
            sub_l_cont_f_ins = o_fragArray->sub_l_cont[id_f_ins];              // UDPATE
            l_cont_bp_f_ins = o_fragArray->l_cont_bp[id_f_ins];
            len_bp_f_ins = o_fragArray->len_bp[id_f_ins];
            sub_len_f_ins = o_fragArray->sub_len[id_f_ins];              // UDPATE
            start_bp_f_ins = o_fragArray->start_bp[id_f_ins];
            id_prev_f_ins = o_fragArray->prev[id_f_ins];
            id_next_f_ins = o_fragArray->next[id_f_ins];
            circ_f_ins = o_fragArray->circ[id_f_ins];
            or_f_ins = o_fragArray->ori[id_f_ins];
            activ_f_ins = o_fragArray->activ[id_f_ins];
        }
        __syncthreads();


        if ((activ_f_ins == 1) && ( activ_f_pop == 1)) {
            if (id_frag == id_f_pop) {
                fragArray->pos[id_frag] = 0;
                fragArray->sub_pos[id_frag] = 0;              // UDPATE
                fragArray->start_bp[id_frag] = 0;
                fragArray->len_bp[id_frag] = len_bp_f_pop;
                fragArray->sub_len[id_frag] = sub_len_f_pop;              // UDPATE
                fragArray->circ[id_frag] = 0;
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = ori_f_pop;
                fragArray->prev[id_frag] = -1;
                fragArray->next[id_frag] = id_f_ins;
                if (circ_f_ins == 0) {
                    fragArray->id_c[id_frag] = max_id_contig + 1;
                    fragArray->l_cont[id_frag] = l_cont_f_ins - pos_f_ins + 1;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins - start_bp_f_ins + len_bp_f_pop;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins - sub_pos_f_ins + sub_len_f_pop;              // UDPATE
                }
                else {
                    fragArray->id_c[id_frag] = contig_f_ins;
                    fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;
                }
                fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
            }
            else if ((id_frag < n_frags) && (id_frag != id_f_pop)) {
                int contig_fi = o_fragArray->id_c[id_frag];
                int pos_fi = o_fragArray->pos[id_frag];
                int sub_pos_fi = o_fragArray->sub_pos[id_frag];              // UDPATE
                int l_cont_fi = o_fragArray->l_cont[id_frag];
                int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag];              // UDPATE
                int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
                int len_bp_fi = o_fragArray->len_bp[id_frag];
                int sub_len_fi = o_fragArray->sub_len[id_frag];              // UDPATE
                int circ_fi = o_fragArray->circ[id_frag];
                int id_prev_fi = o_fragArray->prev[id_frag];
                int id_next_fi = o_fragArray->next[id_frag];
                int start_bp_fi = o_fragArray->start_bp[id_frag];
                int or_fi = o_fragArray->ori[id_frag];
                int rep_fi = o_fragArray->rep[id_frag];
                int activ_fi = o_fragArray->activ[id_frag];
                int id_d_fi = o_fragArray->id_d[id_frag];

                if (contig_fi == contig_f_ins) {
                    if (circ_f_ins == 0) {
                        if (pos_fi < pos_f_ins) {
                            fragArray->pos[id_frag] = pos_fi;
                            fragArray->sub_pos[id_frag] = sub_pos_fi; // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->prev[id_frag] = id_prev_fi;
                            if (pos_fi == (pos_f_ins -1)) {
                                fragArray->next[id_frag] = -1;
                            }
                            else {
                                fragArray->next[id_frag] = id_next_fi;
                            }
                            fragArray->l_cont[id_frag] = pos_f_ins;
                            fragArray->sub_l_cont[id_frag] = sub_pos_f_ins;  // UPDATE
                            fragArray->l_cont_bp[id_frag] = start_bp_f_ins;
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;
                        }
                        else if (pos_fi == pos_f_ins) {
                            fragArray->pos[id_frag] = 1;
                            fragArray->sub_pos[id_frag] = sub_len_f_pop;  // UPDATE
                            fragArray->id_c[id_frag] = max_id_contig + 1;
                            fragArray->start_bp[id_frag] = len_bp_f_pop;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_f_ins;
                            fragArray->prev[id_frag] = id_f_pop;
                            fragArray->next[id_frag] = id_next_f_ins;
                            fragArray->l_cont[id_frag] = l_cont_f_ins - pos_f_ins + 1;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins - sub_pos_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins - start_bp_f_ins + len_bp_f_pop;
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;
                        }
                        else if (pos_fi > pos_f_ins) {
                            fragArray->pos[id_frag] = pos_fi - (pos_f_ins) + 1;
                            fragArray->sub_pos[id_frag] = sub_pos_fi - (sub_pos_f_ins) + sub_len_f_pop;  // UPDATE
                            fragArray->id_c[id_frag] = max_id_contig + 1;
                            fragArray->start_bp[id_frag] = start_bp_fi - (start_bp_f_ins) + len_bp_f_pop;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->prev[id_frag] = id_prev_fi;
                            fragArray->next[id_frag] = id_next_fi;
                            fragArray->l_cont[id_frag] = l_cont_f_ins - pos_f_ins + 1;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins - start_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins - sub_pos_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;
                        }
                    }
                    else { // contig_f_ins is circular
                        if (pos_fi < pos_f_ins) {
                            fragArray->pos[id_frag] = l_cont_f_ins - pos_f_ins + pos_fi + 1;
                            fragArray->sub_pos[id_frag] = sub_l_cont_f_ins - sub_pos_f_ins + sub_pos_fi + sub_len_f_pop;  // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = l_cont_bp_f_ins - start_bp_f_ins + start_bp_fi + len_bp_f_pop;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->prev[id_frag] = id_prev_fi;
                            if (pos_fi == pos_f_ins - 1) {
                                fragArray->next[id_frag] = -1;
                            }
                            else {
                                fragArray->next[id_frag] = id_next_fi;
                            }
                            fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;
                        }
                        else if (pos_fi == pos_f_ins) {
                            fragArray->pos[id_frag] = 1;
                            fragArray->sub_pos[id_frag] = sub_len_f_pop; // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = len_bp_f_pop;
                            fragArray->len_bp[id_frag] = len_bp_f_ins;
                            fragArray->sub_len[id_frag] = sub_len_f_ins; // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_f_ins;
                            fragArray->prev[id_frag] = id_f_pop;
                            fragArray->next[id_frag] = id_next_f_ins;
                            fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;
                        }
                        else if (pos_fi > pos_f_ins) {
                            fragArray->pos[id_frag] = pos_fi - pos_f_ins + 1;
                            fragArray->sub_pos[id_frag] = sub_pos_fi - sub_pos_f_ins + sub_len_f_pop; // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = start_bp_fi - start_bp_f_ins + len_bp_f_pop;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->prev[id_frag] = id_prev_fi;
                            if (id_frag == id_prev_f_ins) {
                                fragArray->next[id_frag] = -1;
                            }
                            else {
                                fragArray->next[id_frag] = id_next_fi;
                            }
//                            fragArray->next[id_frag] = id_next_fi;
                            fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;
                        }
                    }
                }
                else {
                    fragArray->pos[id_frag] = pos_fi;
                    fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                    fragArray->id_c[id_frag] = contig_fi;
                    fragArray->start_bp[id_frag] = start_bp_fi;
                    fragArray->len_bp[id_frag] = len_bp_fi;
                    fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                    fragArray->circ[id_frag] = circ_fi;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = or_fi;
                    fragArray->prev[id_frag] = id_prev_fi;
                    fragArray->next[id_frag] = id_next_fi;
                    fragArray->l_cont[id_frag] = l_cont_fi;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;  // UPDATE
                    fragArray->rep[id_frag] = rep_fi;
                    fragArray->activ[id_frag] = activ_fi;
                    fragArray->id_d[id_frag] = id_d_fi;
                }
            }
        }
        else {
            if (id_frag < n_frags) {
                fragArray->pos[id_frag] = o_fragArray->pos[id_frag];
                fragArray->sub_pos[id_frag] = o_fragArray->sub_pos[id_frag];  // UPDATE
                fragArray->id_c[id_frag] = o_fragArray->id_c[id_frag];
                fragArray->circ[id_frag] = o_fragArray->circ[id_frag];
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = o_fragArray->ori[id_frag];
                fragArray->start_bp[id_frag] = o_fragArray->start_bp[id_frag];
                fragArray->len_bp[id_frag] = o_fragArray->len_bp[id_frag];
                fragArray->sub_len[id_frag] = o_fragArray->sub_len[id_frag];  // UPDATE
                fragArray->prev[id_frag] = o_fragArray->prev[id_frag];
                fragArray->next[id_frag] = o_fragArray->next[id_frag];
                fragArray->l_cont[id_frag] = o_fragArray->l_cont[id_frag];
                fragArray->l_cont_bp[id_frag] = o_fragArray->l_cont_bp[id_frag];
                fragArray->sub_l_cont[id_frag] = o_fragArray->sub_l_cont[id_frag];  // UPDATE
                fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
            }
        }
    }

    __global__ void pop_in_frag_2(frag* fragArray,frag* o_fragArray, int id_f_pop, int id_f_ins, int max_id_contig,
                                  int ori_f_pop,
                                  int n_frags)
    {
        // split insert @ right
        __shared__ int contig_f_pop;
        __shared__ int pos_f_pop;
        __shared__ int sub_pos_f_pop;  // UPDATE
        __shared__ int l_cont_f_pop;
        __shared__ int l_cont_bp_f_pop;
        __shared__ int sub_l_cont_f_pop;  // UPDATE
        __shared__ int len_bp_f_pop;
        __shared__ int sub_len_f_pop;  // UPDATE
        __shared__ int start_bp_f_pop;
        __shared__ int id_prev_f_pop;
        __shared__ int id_next_f_pop;
        __shared__ int activ_f_pop;
//        __shared__ int or_f_pop;

        __shared__ int contig_f_ins;
        __shared__ int pos_f_ins;
        __shared__ int sub_pos_f_ins; // UPDATE
        __shared__ int l_cont_f_ins;
        __shared__ int l_cont_bp_f_ins;
        __shared__ int sub_l_cont_f_ins;  // UPDATE
        __shared__ int len_bp_f_ins;
        __shared__ int sub_len_f_ins;  // UPDATE
        __shared__ int start_bp_f_ins;
        __shared__ int id_prev_f_ins;
        __shared__ int id_next_f_ins;
        __shared__ int circ_f_ins;
        __shared__ int or_f_ins;
        __shared__ int activ_f_ins;

        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            contig_f_pop = o_fragArray->id_c[id_f_pop];
            pos_f_pop = o_fragArray->pos[id_f_pop];
            sub_pos_f_pop = o_fragArray->sub_pos[id_f_pop];  // UPDATE
            l_cont_f_pop = o_fragArray->l_cont[id_f_pop];
            l_cont_bp_f_pop = o_fragArray->l_cont_bp[id_f_pop];
            sub_l_cont_f_pop = o_fragArray->sub_l_cont[id_f_pop];  // UPDATE
            len_bp_f_pop = o_fragArray->len_bp[id_f_pop];
            sub_len_f_pop = o_fragArray->sub_len[id_f_pop];  // UPDATE
            start_bp_f_pop = o_fragArray->start_bp[id_f_pop];
            id_prev_f_pop = o_fragArray->prev[id_f_pop];
            id_next_f_pop = o_fragArray->next[id_f_pop];
            activ_f_pop = o_fragArray->activ[id_f_pop];

            contig_f_ins = o_fragArray->id_c[id_f_ins];
            pos_f_ins = o_fragArray->pos[id_f_ins];
            sub_pos_f_ins = o_fragArray->sub_pos[id_f_ins];  // UPDATE
            l_cont_f_ins = o_fragArray->l_cont[id_f_ins];
            l_cont_bp_f_ins = o_fragArray->l_cont_bp[id_f_ins];
            sub_l_cont_f_ins = o_fragArray->sub_l_cont[id_f_ins];  // UPDATE
            len_bp_f_ins = o_fragArray->len_bp[id_f_ins];
            sub_len_f_ins = o_fragArray->sub_len[id_f_ins];  // UPDATE
            start_bp_f_ins = o_fragArray->start_bp[id_f_ins];
            id_prev_f_ins = o_fragArray->prev[id_f_ins];
            id_next_f_ins = o_fragArray->next[id_f_ins];
            circ_f_ins = o_fragArray->circ[id_f_ins];
            or_f_ins = o_fragArray->ori[id_f_ins];
            activ_f_ins = o_fragArray->activ[id_f_ins];
        }
        __syncthreads();
        if ((activ_f_ins == 1) && ( activ_f_pop == 1)) {
            if (id_frag == id_f_pop) {
                if (circ_f_ins == 0) {
                    fragArray->pos[id_frag] = pos_f_ins + 1;
                    fragArray->sub_pos[id_frag] = sub_pos_f_ins + sub_len_f_ins;  // UPDATE
                    fragArray->id_c[id_frag] = contig_f_ins;
                    fragArray->start_bp[id_frag] = start_bp_f_ins + len_bp_f_ins;
                    fragArray->len_bp[id_frag] = len_bp_f_pop;
                    fragArray->sub_len[id_frag] = sub_len_f_pop;  // UPDATE
                    fragArray->circ[id_frag] = 0;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = ori_f_pop;
                    fragArray->prev[id_frag] = id_f_ins;
                    fragArray->next[id_frag] = -1;
                    fragArray->l_cont[id_frag] = pos_f_ins + 2;
                    fragArray->l_cont_bp[id_frag] = start_bp_f_ins + len_bp_f_ins + len_bp_f_pop;
                    fragArray->sub_l_cont[id_frag] = sub_pos_f_ins + sub_len_f_ins + sub_len_f_pop;  // UPDATE
                    fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                    fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                    fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
                }
                else {
                    fragArray->pos[id_frag] = (l_cont_f_ins - (pos_f_ins  + 1)) + pos_f_ins + 1;
                    fragArray->sub_pos[id_frag] = (sub_l_cont_f_ins - (sub_pos_f_ins  + sub_len_f_ins))
                                                  + sub_pos_f_ins + sub_len_f_ins;  // UPDATE
                    fragArray->id_c[id_frag] = contig_f_ins;
                    fragArray->start_bp[id_frag] = (l_cont_bp_f_ins - (start_bp_f_ins + len_bp_f_ins))
                                                   + start_bp_f_ins + len_bp_f_ins;
                    fragArray->len_bp[id_frag] = len_bp_f_pop;
                    fragArray->sub_len[id_frag] = sub_len_f_pop;  // UPDATE
                    fragArray->circ[id_frag] = 0;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = ori_f_pop;
                    fragArray->prev[id_frag] = id_f_ins;
                    fragArray->next[id_frag] = -1;
                    fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                    fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                    fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                    fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
                }
            }
            else if ((id_frag < n_frags) && (id_frag != id_f_pop)) {
                int contig_fi = o_fragArray->id_c[id_frag];
                int pos_fi = o_fragArray->pos[id_frag];
                int sub_pos_fi = o_fragArray->sub_pos[id_frag];  // UPDATE
                int l_cont_fi = o_fragArray->l_cont[id_frag];
                int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
                int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag]; // UPDATE
                int len_bp_fi = o_fragArray->len_bp[id_frag];
                int sub_len_fi = o_fragArray->sub_len[id_frag]; // UPDATE
                int circ_fi = o_fragArray->circ[id_frag];
                int id_prev_fi = o_fragArray->prev[id_frag];
                int id_next_fi = o_fragArray->next[id_frag];
                int start_bp_fi = o_fragArray->start_bp[id_frag];
                int or_fi = o_fragArray->ori[id_frag];
                int rep_fi = o_fragArray->rep[id_frag];
                int activ_fi = o_fragArray->activ[id_frag];
                int id_d_fi = o_fragArray->id_d[id_frag];
                if (contig_fi == contig_f_ins) {
                    if(circ_f_ins == 0) {
                        if (pos_fi < pos_f_ins) {
                            fragArray->pos[id_frag] = pos_fi;
                            fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->prev[id_frag] = id_prev_fi;
                            fragArray->next[id_frag] = id_next_fi;
                            fragArray->l_cont[id_frag] = pos_f_ins + 2;
                            fragArray->l_cont_bp[id_frag] = start_bp_f_ins + len_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_pos_f_ins + sub_len_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;
                        }
                        else if (pos_fi == pos_f_ins) {
                            fragArray->pos[id_frag] = pos_fi;
                            fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_f_ins;
                            fragArray->prev[id_frag] = id_prev_f_ins;
                            fragArray->next[id_frag] = id_f_pop;
                            fragArray->l_cont[id_frag] = pos_f_ins + 2;
                            fragArray->l_cont_bp[id_frag] = start_bp_f_ins + len_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_pos_f_ins + sub_len_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;

                        }
                        else if (pos_fi > pos_f_ins) {
                            fragArray->pos[id_frag] = pos_fi - (pos_f_ins + 1);
                            fragArray->sub_pos[id_frag] = sub_pos_fi - (sub_pos_f_ins + sub_len_f_ins);  // UPDATE
                            fragArray->id_c[id_frag] = max_id_contig + 1;
                            fragArray->start_bp[id_frag] = start_bp_fi - (start_bp_f_ins + len_bp_f_ins);
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            if (pos_fi == (pos_f_ins + 1)) {
                                fragArray->prev[id_frag] = -1;
                            }
                            else {
                                fragArray->prev[id_frag] = id_prev_fi;
                            }
                            fragArray->next[id_frag] = id_next_fi;
                            fragArray->l_cont[id_frag] = l_cont_f_ins - (pos_f_ins + 1);
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins - (start_bp_f_ins + len_bp_f_ins);
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins - (sub_pos_f_ins + sub_len_f_ins);  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;

                        }
                    }
                    else { //circular contig
                        if (pos_fi < pos_f_ins) {
                            fragArray->pos[id_frag] = (l_cont_f_ins - (pos_f_ins  + 1)) + pos_fi;
                            fragArray->sub_pos[id_frag] = (sub_l_cont_f_ins - (sub_pos_f_ins  + sub_len_f_ins))
                                                          + sub_pos_fi;  // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = (l_cont_bp_f_ins - (start_bp_f_ins + len_bp_f_ins))
                                                           + start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            if (id_frag == id_next_f_ins) {
                                fragArray->prev[id_frag] = -1;
                            }
                            else {
                                fragArray->prev[id_frag] = id_prev_fi;
                            }
//                            fragArray->prev[id_frag] = id_prev_fi;
                            fragArray->next[id_frag] = id_next_fi;
                            fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;

                        }
                        else if (pos_fi == pos_f_ins) {
                            fragArray->pos[id_frag] = (l_cont_f_ins - (pos_f_ins  + 1)) + pos_f_ins;
                            fragArray->sub_pos[id_frag] = (sub_l_cont_f_ins - (sub_pos_f_ins  + sub_len_f_ins))
                                                          + sub_pos_f_ins;  // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = (l_cont_bp_f_ins - (start_bp_f_ins + len_bp_f_ins))
                                                           + start_bp_f_ins;
                            fragArray->len_bp[id_frag] = len_bp_f_ins;
                            fragArray->sub_len[id_frag] = sub_len_f_ins;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->prev[id_frag] = id_prev_f_ins;
                            fragArray->next[id_frag] = id_f_pop;
                            fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;

                        }
                        else if (pos_fi > pos_f_ins) {
                            fragArray->pos[id_frag] = pos_fi - (pos_f_ins + 1);
                            fragArray->sub_pos[id_frag] = sub_pos_fi - (sub_pos_f_ins + sub_len_f_ins);  // UPDATE
                            fragArray->id_c[id_frag] = contig_f_ins;
                            fragArray->start_bp[id_frag] = start_bp_fi - (start_bp_f_ins + len_bp_f_ins);
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            if (pos_fi == pos_f_ins +1) {
                                fragArray->prev[id_frag] = -1;
                            }
                            else {
                                fragArray->prev[id_frag] = id_prev_fi;
                            }
                            fragArray->next[id_frag] = id_next_fi;
                            fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                            fragArray->rep[id_frag] = rep_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->id_d[id_frag] = id_d_fi;

                        }
                    }
                }
                else {
                    fragArray->pos[id_frag] = pos_fi;
                    fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                    fragArray->id_c[id_frag] = contig_fi;
                    fragArray->start_bp[id_frag] = start_bp_fi;
                    fragArray->len_bp[id_frag] = len_bp_fi;
                    fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                    fragArray->circ[id_frag] = circ_fi;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = or_fi;
                    fragArray->prev[id_frag] = id_prev_fi;
                    fragArray->next[id_frag] = id_next_fi;
                    fragArray->l_cont[id_frag] = l_cont_fi;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;  // UPDATE
                    fragArray->rep[id_frag] = rep_fi;
                    fragArray->activ[id_frag] = activ_fi;
                    fragArray->id_d[id_frag] = id_d_fi;

                }
            }
        }
        else {
            if (id_frag < n_frags) {
                fragArray->pos[id_frag] = o_fragArray->pos[id_frag];
                fragArray->sub_pos[id_frag] = o_fragArray->sub_pos[id_frag];  // UPDATE
                fragArray->id_c[id_frag] = o_fragArray->id_c[id_frag];
                fragArray->circ[id_frag] = o_fragArray->circ[id_frag];
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = o_fragArray->ori[id_frag];
                fragArray->start_bp[id_frag] = o_fragArray->start_bp[id_frag];
                fragArray->len_bp[id_frag] = o_fragArray->len_bp[id_frag];
                fragArray->sub_len[id_frag] = o_fragArray->sub_len[id_frag];  // UPDATE
                fragArray->prev[id_frag] = o_fragArray->prev[id_frag];
                fragArray->next[id_frag] = o_fragArray->next[id_frag];
                fragArray->l_cont[id_frag] = o_fragArray->l_cont[id_frag];
                fragArray->l_cont_bp[id_frag] = o_fragArray->l_cont_bp[id_frag];
                fragArray->sub_l_cont[id_frag] = o_fragArray->sub_l_cont[id_frag];  // UPDATE
                fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
            }
        }
    }

    __global__ void pop_in_frag_3(frag* fragArray,frag* o_fragArray, int id_f_pop, int id_f_ins, int max_id_contig,
                                  int ori_f_pop,
                                  int n_frags)
    // insert frag @ right of id_f_ins
    {
        __shared__ int contig_f_pop;
        __shared__ int pos_f_pop;
        __shared__ int sub_pos_f_pop;  // UPDATE
        __shared__ int l_cont_f_pop;
        __shared__ int l_cont_bp_f_pop;
        __shared__ int sub_l_cont_f_pop;  // UPDATE
        __shared__ int len_bp_f_pop;
        __shared__ int sub_len_f_pop;  // UPDATE
        __shared__ int start_bp_f_pop;
        __shared__ int id_prev_f_pop;
        __shared__ int id_next_f_pop;
        __shared__ int activ_f_pop;
//        __shared__ int or_f_pop;

        __shared__ int contig_f_ins;
        __shared__ int pos_f_ins;
        __shared__ int sub_pos_f_ins;  // UPDATE
        __shared__ int l_cont_f_ins;
        __shared__ int l_cont_bp_f_ins;
        __shared__ int sub_l_cont_f_ins;  // UPDATE
        __shared__ int len_bp_f_ins;
        __shared__ int sub_len_f_ins;  // UPDATE
        __shared__ int start_bp_f_ins;
        __shared__ int id_prev_f_ins;
        __shared__ int id_next_f_ins;
        __shared__ int circ_f_ins;
        __shared__ int or_f_ins;
        __shared__ int activ_f_ins;

        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            contig_f_pop = o_fragArray->id_c[id_f_pop];
            pos_f_pop = o_fragArray->pos[id_f_pop];
            sub_pos_f_pop = o_fragArray->sub_pos[id_f_pop];  // UPDATE
            l_cont_f_pop = o_fragArray->l_cont[id_f_pop];
            l_cont_bp_f_pop = o_fragArray->l_cont_bp[id_f_pop];
            sub_l_cont_f_pop = o_fragArray->sub_l_cont[id_f_pop];  // UPDATE
            len_bp_f_pop = o_fragArray->len_bp[id_f_pop];
            sub_len_f_pop = o_fragArray->sub_len[id_f_pop];  // UPDATE
            start_bp_f_pop = o_fragArray->start_bp[id_f_pop];
            id_prev_f_pop = o_fragArray->prev[id_f_pop];
            id_next_f_pop = o_fragArray->next[id_f_pop];
            activ_f_pop = o_fragArray->activ[id_f_pop];

            contig_f_ins = o_fragArray->id_c[id_f_ins];
            pos_f_ins = o_fragArray->pos[id_f_ins];
            sub_pos_f_ins = o_fragArray->sub_pos[id_f_ins];  // UPDATE
            l_cont_f_ins = o_fragArray->l_cont[id_f_ins];
            l_cont_bp_f_ins = o_fragArray->l_cont_bp[id_f_ins];
            sub_l_cont_f_ins = o_fragArray->sub_l_cont[id_f_ins];  // UPDATE
            len_bp_f_ins = o_fragArray->len_bp[id_f_ins];
            sub_len_f_ins = o_fragArray->sub_len[id_f_ins];  // UPDATE
            start_bp_f_ins = o_fragArray->start_bp[id_f_ins];
            id_prev_f_ins = o_fragArray->prev[id_f_ins];
            id_next_f_ins = o_fragArray->next[id_f_ins];
            circ_f_ins = o_fragArray->circ[id_f_ins];
            or_f_ins = o_fragArray->ori[id_f_ins];
            activ_f_ins = o_fragArray->activ[id_f_ins];
        }
        __syncthreads();
        if ((activ_f_ins == 1) && ( activ_f_pop == 1)) {
            if (id_frag == id_f_pop) {
                fragArray->pos[id_frag] = pos_f_ins + 1;
                fragArray->sub_pos[id_frag] = sub_pos_f_ins + sub_len_f_ins;  // UPDATE
                fragArray->id_c[id_frag] = contig_f_ins;
                fragArray->start_bp[id_frag] = start_bp_f_ins + len_bp_f_ins;
                fragArray->len_bp[id_frag] = len_bp_f_pop;
                fragArray->sub_len[id_frag] = sub_len_f_pop;  // UPDATE
                fragArray->circ[id_frag] = circ_f_ins;
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = ori_f_pop;
                fragArray->prev[id_frag] = id_f_ins;
                fragArray->next[id_frag] = id_next_f_ins;
                fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
            }
            else if ((id_frag < n_frags) && (id_frag != id_f_pop)) {
                int contig_fi = o_fragArray->id_c[id_frag];
                int pos_fi = o_fragArray->pos[id_frag];
                int sub_pos_fi = o_fragArray->sub_pos[id_frag];  // UPDATE
                int l_cont_fi = o_fragArray->l_cont[id_frag];
                int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
                int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag];  // UPDATE
                int len_bp_fi = o_fragArray->len_bp[id_frag];
                int sub_len_fi = o_fragArray->sub_len[id_frag];  // UPDATE
                int circ_fi = o_fragArray->circ[id_frag];
                int id_prev_fi = o_fragArray->prev[id_frag];
                int id_next_fi = o_fragArray->next[id_frag];
                int start_bp_fi = o_fragArray->start_bp[id_frag];
                int or_fi = o_fragArray->ori[id_frag];
                int rep_fi = o_fragArray->rep[id_frag];
                int activ_fi = o_fragArray->activ[id_frag];
                int id_d_fi = o_fragArray->id_d[id_frag];
                if (contig_fi == contig_f_ins) {
                    if (pos_fi < pos_f_ins) {
                        fragArray->pos[id_frag] = pos_fi;
                        fragArray->sub_pos[id_frag] = sub_pos_fi; // UPDATE
                        fragArray->id_c[id_frag] = contig_f_ins;
                        fragArray->start_bp[id_frag] = start_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                        fragArray->circ[id_frag] = circ_f_ins;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        if ((id_frag == id_next_f_ins) && ( circ_f_ins == 1)) {
                            fragArray->prev[id_frag] = id_f_pop;
                        }
                        else {
                            fragArray->prev[id_frag] = id_prev_fi;
                        }
                        fragArray->next[id_frag] = id_next_fi;
                        fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop; // UPDATE
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;

                    }
                    else if (pos_fi == pos_f_ins) {
                        fragArray->pos[id_frag] = pos_fi;
                        fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                        fragArray->id_c[id_frag] = contig_f_ins;
                        fragArray->start_bp[id_frag] = start_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                        fragArray->circ[id_frag] = circ_f_ins;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_f_ins;
                        fragArray->prev[id_frag] = id_prev_fi;
                        fragArray->next[id_frag] = id_f_pop;
                        fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;

                    }
                    else if (pos_fi > pos_f_ins) {
                        fragArray->pos[id_frag] = pos_fi + 1;
                        fragArray->sub_pos[id_frag] = sub_pos_fi + sub_len_f_pop;  // UPDATE
                        fragArray->id_c[id_frag] = contig_f_ins;
                        fragArray->start_bp[id_frag] = start_bp_fi + len_bp_f_pop;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                        fragArray->circ[id_frag] = circ_f_ins;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        if (pos_fi == (pos_f_ins + 1)) {
                            fragArray->prev[id_frag] = id_f_pop;
                        }
                        else {
                            fragArray->prev[id_frag] = id_prev_fi;
                        }
                        fragArray->next[id_frag] = id_next_fi;
                        fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                        fragArray->rep[id_frag] = rep_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->id_d[id_frag] = id_d_fi;

                    }
                }
                else {
                    fragArray->pos[id_frag] = pos_fi;
                    fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                    fragArray->id_c[id_frag] = contig_fi;
                    fragArray->start_bp[id_frag] = start_bp_fi;
                    fragArray->len_bp[id_frag] = len_bp_fi;
                    fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                    fragArray->circ[id_frag] = circ_fi;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = or_fi;
                    fragArray->prev[id_frag] = id_prev_fi;
                    fragArray->next[id_frag] = id_next_fi;
                    fragArray->l_cont[id_frag] = l_cont_fi;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;  // UPDATE
                    fragArray->id_d[id_frag] = id_d_fi;
                    fragArray->activ[id_frag] = activ_fi;
                    fragArray->rep[id_frag] = rep_fi;

                }
            }
        }
        else {
            if (id_frag < n_frags) {
                fragArray->pos[id_frag] = o_fragArray->pos[id_frag];
                fragArray->sub_pos[id_frag] = o_fragArray->sub_pos[id_frag];  // UPDATE
                fragArray->id_c[id_frag] = o_fragArray->id_c[id_frag];
                fragArray->circ[id_frag] = o_fragArray->circ[id_frag];
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = o_fragArray->ori[id_frag];
                fragArray->start_bp[id_frag] = o_fragArray->start_bp[id_frag];
                fragArray->len_bp[id_frag] = o_fragArray->len_bp[id_frag];
                fragArray->sub_len[id_frag] = o_fragArray->sub_len[id_frag];  // UPDATE
                fragArray->prev[id_frag] = o_fragArray->prev[id_frag];
                fragArray->next[id_frag] = o_fragArray->next[id_frag];
                fragArray->l_cont[id_frag] = o_fragArray->l_cont[id_frag];
                fragArray->l_cont_bp[id_frag] = o_fragArray->l_cont_bp[id_frag];
                fragArray->sub_l_cont[id_frag] = o_fragArray->sub_l_cont[id_frag];  // UPDATE
                fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
            }
        }
    }

    __global__ void pop_in_frag_4(frag* fragArray,frag* o_fragArray, int id_f_pop, int id_f_ins, int max_id_contig,
                                  int ori_f_pop,
                                  int n_frags)
    // insert frag @ left of id_f_ins
    {
        __shared__ int contig_f_pop;
        __shared__ int pos_f_pop;
        __shared__ int sub_pos_f_pop;  // UPDATE
        __shared__ int l_cont_f_pop;
        __shared__ int l_cont_bp_f_pop;
        __shared__ int sub_l_cont_f_pop;  // UPDATE
        __shared__ int len_bp_f_pop;
        __shared__ int sub_len_f_pop;  // UPDATE
        __shared__ int start_bp_f_pop;
        __shared__ int id_prev_f_pop;
        __shared__ int id_next_f_pop;
        __shared__ int activ_f_pop;
//        __shared__ int or_f_pop;

        __shared__ int contig_f_ins;
        __shared__ int pos_f_ins;
        __shared__ int sub_pos_f_ins;  // UPDATE
        __shared__ int l_cont_f_ins;
        __shared__ int l_cont_bp_f_ins;
        __shared__ int sub_l_cont_f_ins;  // UPDATE
        __shared__ int len_bp_f_ins;
        __shared__ int sub_len_f_ins;  // UPDATE
        __shared__ int start_bp_f_ins;
        __shared__ int id_prev_f_ins;
        __shared__ int id_next_f_ins;
        __shared__ int circ_f_ins;
        __shared__ int or_f_ins;
        __shared__ int activ_f_ins;

        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            contig_f_pop = o_fragArray->id_c[id_f_pop];
            pos_f_pop = o_fragArray->pos[id_f_pop];
            sub_pos_f_pop = o_fragArray->sub_pos[id_f_pop];  // UPDATE
            l_cont_f_pop = o_fragArray->l_cont[id_f_pop];
            l_cont_bp_f_pop = o_fragArray->l_cont_bp[id_f_pop];
            sub_l_cont_f_pop = o_fragArray->sub_l_cont[id_f_pop];  // UPDATE
            len_bp_f_pop = o_fragArray->len_bp[id_f_pop];
            sub_len_f_pop = o_fragArray->sub_len[id_f_pop];  // UPDATE
            start_bp_f_pop = o_fragArray->start_bp[id_f_pop];
            id_prev_f_pop = o_fragArray->prev[id_f_pop];
            id_next_f_pop = o_fragArray->next[id_f_pop];
            activ_f_pop = o_fragArray->activ[id_f_pop];

            contig_f_ins = o_fragArray->id_c[id_f_ins];
            pos_f_ins = o_fragArray->pos[id_f_ins];
            sub_pos_f_ins = o_fragArray->sub_pos[id_f_ins];  // UPDATE
            l_cont_f_ins = o_fragArray->l_cont[id_f_ins];
            l_cont_bp_f_ins = o_fragArray->l_cont_bp[id_f_ins];
            sub_l_cont_f_ins = o_fragArray->sub_l_cont[id_f_ins];  // UPDATE
            len_bp_f_ins = o_fragArray->len_bp[id_f_ins];
            sub_len_f_ins = o_fragArray->sub_len[id_f_ins];  // UPDATE
            start_bp_f_ins = o_fragArray->start_bp[id_f_ins];
            id_prev_f_ins = o_fragArray->prev[id_f_ins];
            id_next_f_ins = o_fragArray->next[id_f_ins];
            circ_f_ins = o_fragArray->circ[id_f_ins];
            or_f_ins = o_fragArray->ori[id_f_ins];
            activ_f_ins = o_fragArray->activ[id_f_ins];
        }
        __syncthreads();
        if ((activ_f_ins == 1) && ( activ_f_pop == 1)) {
            if (id_frag == id_f_pop) {
                fragArray->pos[id_frag] = pos_f_ins ;
                fragArray->sub_pos[id_frag] = sub_pos_f_ins ;  // UPDATE
                fragArray->id_c[id_frag] = contig_f_ins;
                fragArray->start_bp[id_frag] = start_bp_f_ins;
                fragArray->len_bp[id_frag] = len_bp_f_pop;
                fragArray->sub_len[id_frag] = sub_len_f_pop;  // UPDATE
                fragArray->circ[id_frag] = circ_f_ins;
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = ori_f_pop;
                fragArray->prev[id_frag] = id_prev_f_ins;
                fragArray->next[id_frag] = id_f_ins;
                fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];

            }
            else if ((id_frag < n_frags) && (id_frag != id_f_pop)) {
                int contig_fi = o_fragArray->id_c[id_frag];
                int pos_fi = o_fragArray->pos[id_frag];
                int sub_pos_fi = o_fragArray->sub_pos[id_frag];  // UPDATE
                int l_cont_fi = o_fragArray->l_cont[id_frag];
                int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
                int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag];  // UPDATE
                int len_bp_fi = o_fragArray->len_bp[id_frag];
                int sub_len_fi = o_fragArray->sub_len[id_frag];  // UPDATE
                int circ_fi = o_fragArray->circ[id_frag];
                int id_prev_fi = o_fragArray->prev[id_frag];
                int id_next_fi = o_fragArray->next[id_frag];
                int start_bp_fi = o_fragArray->start_bp[id_frag];
                int or_fi = o_fragArray->ori[id_frag];
                int rep_fi = o_fragArray->rep[id_frag];
                int activ_fi = o_fragArray->activ[id_frag];
                int id_d_fi = o_fragArray->id_d[id_frag];

                if (contig_fi == contig_f_ins) {
                    if (pos_fi < pos_f_ins) {
                        fragArray->pos[id_frag] = pos_fi;
                        fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                        fragArray->id_c[id_frag] = contig_f_ins;
                        fragArray->start_bp[id_frag] = start_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                        fragArray->circ[id_frag] = circ_f_ins;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        fragArray->prev[id_frag] = id_prev_fi;
                        if (pos_fi == pos_f_ins -1) {
                            fragArray->next[id_frag] = id_f_pop;
                        }
                        else {
                            fragArray->next[id_frag] = id_next_fi;
                        }
                        fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                        fragArray->id_d[id_frag] = id_d_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->rep[id_frag] = rep_fi;

                    }
                    else if (pos_fi == pos_f_ins) {
                        fragArray->pos[id_frag] = pos_f_ins + 1;
                        fragArray->sub_pos[id_frag] = sub_pos_f_ins + sub_len_f_pop;  // UPDATE
                        fragArray->id_c[id_frag] = contig_f_ins;
                        fragArray->start_bp[id_frag] = start_bp_f_ins + len_bp_f_pop;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                        fragArray->circ[id_frag] = circ_f_ins;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_f_ins;
                        fragArray->prev[id_frag] = id_f_pop;
                        fragArray->next[id_frag] = id_next_f_ins;
                        fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                        fragArray->id_d[id_frag] = id_d_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->rep[id_frag] = rep_fi;

                    }
                    else if (pos_fi > pos_f_ins) {
                        fragArray->pos[id_frag] = pos_fi + 1;
                        fragArray->sub_pos[id_frag] = sub_pos_fi + sub_len_f_pop;  // UPDATE
                        fragArray->id_c[id_frag] = contig_f_ins;
                        fragArray->start_bp[id_frag] = start_bp_fi + len_bp_f_pop;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                        fragArray->circ[id_frag] = circ_f_ins;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        fragArray->prev[id_frag] = id_prev_fi;
                        fragArray->next[id_frag] = id_next_fi;
                        fragArray->l_cont[id_frag] = l_cont_f_ins + 1;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_f_ins + len_bp_f_pop;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_f_ins + sub_len_f_pop;  // UPDATE
                        fragArray->id_d[id_frag] = id_d_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->rep[id_frag] = rep_fi;

                    }
                }
                else {
                    fragArray->pos[id_frag] = pos_fi;
                    fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                    fragArray->id_c[id_frag] = contig_fi;
                    fragArray->start_bp[id_frag] = start_bp_fi;
                    fragArray->len_bp[id_frag] = len_bp_fi;
                    fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                    fragArray->circ[id_frag] = circ_fi;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = or_fi;
                    fragArray->prev[id_frag] = id_prev_fi;
                    fragArray->next[id_frag] = id_next_fi;
                    fragArray->l_cont[id_frag] = l_cont_fi;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;  // UPDATE
                    fragArray->id_d[id_frag] = id_d_fi;
                    fragArray->activ[id_frag] = activ_fi;
                    fragArray->rep[id_frag] = rep_fi;

                }
            }
        }
        else {
            if (id_frag < n_frags) {
                fragArray->pos[id_frag] = o_fragArray->pos[id_frag];
                fragArray->sub_pos[id_frag] = o_fragArray->sub_pos[id_frag];  // UPDATE
                fragArray->id_c[id_frag] = o_fragArray->id_c[id_frag];
                fragArray->circ[id_frag] = o_fragArray->circ[id_frag];
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = o_fragArray->ori[id_frag];
                fragArray->start_bp[id_frag] = o_fragArray->start_bp[id_frag];
                fragArray->len_bp[id_frag] = o_fragArray->len_bp[id_frag];
                fragArray->sub_len[id_frag] = o_fragArray->sub_len[id_frag];  // UPDATE
                fragArray->prev[id_frag] = o_fragArray->prev[id_frag];
                fragArray->next[id_frag] = o_fragArray->next[id_frag];
                fragArray->l_cont[id_frag] = o_fragArray->l_cont[id_frag];
                fragArray->l_cont_bp[id_frag] = o_fragArray->l_cont_bp[id_frag];
                fragArray->sub_l_cont[id_frag] = o_fragArray->sub_l_cont[id_frag];  // UPDATE
                fragArray->rep[id_frag] = o_fragArray->rep[id_frag];
                fragArray->activ[id_frag] = o_fragArray->activ[id_frag];
                fragArray->id_d[id_frag] = o_fragArray->id_d[id_frag];
            }
        }
    }


    __global__ void split_contig(frag* fragArray,frag* o_fragArray, int* split_id_contigs, int id_f_cut, int upstream,
                                 int max_id_contig, int n_frags)
    {
        __shared__ int contig_f_cut;
        __shared__ int pos_f_cut;
        __shared__ int sub_pos_f_cut;  // UPDATE
        __shared__ int l_cont_f_cut;
        __shared__ int l_cont_bp_f_cut;
        __shared__ int sub_l_cont_f_cut;  // UPDATE
        __shared__ int len_bp_f_cut;
        __shared__ int sub_len_f_cut;  // UPDATE
        __shared__ int start_bp_f_cut;
        __shared__ int id_prev_f_cut;
        __shared__ int id_next_f_cut;
        __shared__ int circ_f_cut;
        __shared__ int or_f_cut;
        __shared__ int activ_f_cut;

        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            contig_f_cut = o_fragArray->id_c[id_f_cut];
            pos_f_cut = o_fragArray->pos[id_f_cut];
            sub_pos_f_cut = o_fragArray->sub_pos[id_f_cut];  // UPDATE
            l_cont_f_cut = o_fragArray->l_cont[id_f_cut];
            l_cont_bp_f_cut = o_fragArray->l_cont_bp[id_f_cut];
            sub_l_cont_f_cut = o_fragArray->sub_l_cont[id_f_cut];  // UPDATE
            len_bp_f_cut = o_fragArray->len_bp[id_f_cut];
            sub_len_f_cut = o_fragArray->sub_len[id_f_cut];  // UPDATE
            start_bp_f_cut = o_fragArray->start_bp[id_f_cut];
            id_prev_f_cut = o_fragArray->prev[id_f_cut];
            id_next_f_cut = o_fragArray->next[id_f_cut];
            circ_f_cut = o_fragArray->circ[id_f_cut];
            or_f_cut = o_fragArray->ori[id_f_cut];
            activ_f_cut = o_fragArray->activ[id_f_cut];
        }
        __syncthreads();
        if (id_frag < n_frags) {
            int contig_fi = o_fragArray->id_c[id_frag];
            int pos_fi = o_fragArray->pos[id_frag];
            int sub_pos_fi = o_fragArray->sub_pos[id_frag]; // UPDATE
            int l_cont_fi = o_fragArray->l_cont[id_frag];
            int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
            int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag]; // UPDATE
            int len_bp_fi = o_fragArray->len_bp[id_frag];
            int sub_len_fi = o_fragArray->sub_len[id_frag]; // UPDATE
            int circ_fi = o_fragArray->circ[id_frag];
            int id_prev_fi = o_fragArray->prev[id_frag];
            int id_next_fi = o_fragArray->next[id_frag];
            int start_bp_fi = o_fragArray->start_bp[id_frag];
            int or_fi = o_fragArray->ori[id_frag];
            int rep_fi = o_fragArray->rep[id_frag];
            int activ_fi = o_fragArray->activ[id_frag];
            int id_d_fi = o_fragArray->id_d[id_frag];
            if ((activ_f_cut == 1) && (l_cont_f_cut > 1)) {
                if (contig_fi == contig_f_cut) {
                    if (circ_f_cut == 0) { // linear contig
                        if (upstream == 1) {
                            if (pos_fi < pos_f_cut) {
                                fragArray->pos[id_frag] = pos_fi;
                                fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = start_bp_fi;
                                fragArray->len_bp[id_frag] = len_bp_fi;
                                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = id_prev_fi;
                                if (pos_fi == pos_f_cut - 1) {
                                    fragArray->next[id_frag] = -1;
                                }
                                else {
                                    fragArray->next[id_frag] = id_next_fi;
                                }
                                fragArray->l_cont[id_frag] = pos_f_cut;
                                fragArray->l_cont_bp[id_frag] = start_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_pos_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if(pos_fi == pos_f_cut) {
                                fragArray->pos[id_frag] = 0;
                                fragArray->sub_pos[id_frag] = 0;  // UPDATE
                                fragArray->id_c[id_frag] = max_id_contig + 1;
                                split_id_contigs[id_frag] = max_id_contig + 1;
                                fragArray->start_bp[id_frag] = 0;
                                fragArray->len_bp[id_frag] = len_bp_f_cut;
                                fragArray->sub_len[id_frag] = sub_len_f_cut; // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = -1;
                                fragArray->next[id_frag] = id_next_f_cut;
                                fragArray->l_cont[id_frag] = l_cont_f_cut  - pos_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut - start_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut - sub_pos_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if (pos_fi > pos_f_cut) {
                                fragArray->pos[id_frag] = pos_fi - pos_f_cut;
                                fragArray->sub_pos[id_frag] = sub_pos_fi - sub_pos_f_cut;  // UPDATE
                                fragArray->id_c[id_frag] = max_id_contig + 1;
                                split_id_contigs[id_frag] = max_id_contig + 1;
                                fragArray->start_bp[id_frag] = start_bp_fi - start_bp_f_cut;
                                fragArray->len_bp[id_frag] = len_bp_fi;
                                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = id_prev_fi;
                                fragArray->next[id_frag] = id_next_fi;
                                fragArray->l_cont[id_frag] = l_cont_f_cut  - pos_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut - start_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut - sub_pos_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                        }
                        else {
                            if (pos_fi < pos_f_cut) {
                                fragArray->pos[id_frag] = pos_fi;
                                fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = start_bp_fi;
                                fragArray->len_bp[id_frag] = len_bp_fi;
                                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = id_prev_fi;
                                fragArray->next[id_frag] = id_next_fi;
                                fragArray->l_cont[id_frag] = pos_f_cut + 1;
                                fragArray->l_cont_bp[id_frag] = start_bp_f_cut + len_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_pos_f_cut + sub_len_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if(pos_fi == pos_f_cut) {
                                fragArray->pos[id_frag] = pos_f_cut;
                                fragArray->sub_pos[id_frag] = sub_pos_f_cut;  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = start_bp_f_cut;
                                fragArray->len_bp[id_frag] = len_bp_f_cut;
                                fragArray->sub_len[id_frag] = sub_len_f_cut;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = id_prev_f_cut;
                                fragArray->next[id_frag] = -1;
                                fragArray->l_cont[id_frag] = pos_f_cut + 1;
                                fragArray->l_cont_bp[id_frag] = start_bp_f_cut + len_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_pos_f_cut + sub_len_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if (pos_fi > pos_f_cut) {
                                fragArray->pos[id_frag] = pos_fi - (pos_f_cut + 1);
                                fragArray->sub_pos[id_frag] = sub_pos_fi - (sub_pos_f_cut + sub_len_f_cut);  // UPDATE
                                fragArray->id_c[id_frag] = max_id_contig + 1;
                                split_id_contigs[id_frag] = max_id_contig +1;
                                fragArray->start_bp[id_frag] = start_bp_fi - (start_bp_f_cut + len_bp_f_cut);
                                fragArray->len_bp[id_frag] = len_bp_fi;
                                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                if (pos_fi == pos_f_cut + 1) {
                                    fragArray->prev[id_frag] = -1;
                                }
                                else {
                                    fragArray->prev[id_frag] = id_prev_fi;
                                }
                                fragArray->next[id_frag] = id_next_fi;
                                fragArray->l_cont[id_frag] = l_cont_f_cut - (pos_f_cut + 1);
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut - (start_bp_f_cut + len_bp_f_cut);
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut - (sub_pos_f_cut + sub_len_f_cut);  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                        }
                    }
                    else { // circular contig
                        if (upstream ==1) {
                            if (pos_fi < pos_f_cut) {
                                fragArray->pos[id_frag] = l_cont_f_cut - pos_f_cut + pos_fi;
                                fragArray->sub_pos[id_frag] = sub_l_cont_f_cut - sub_pos_f_cut + sub_pos_fi;  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = l_cont_bp_f_cut - start_bp_f_cut + start_bp_fi;
                                fragArray->len_bp[id_frag] = len_bp_fi;
                                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = id_prev_fi;
                                if (pos_fi == pos_f_cut - 1) {
                                    fragArray->next[id_frag] = -1;
                                }
                                else {
                                    fragArray->next[id_frag] = id_next_fi;
                                }
                                fragArray->l_cont[id_frag] = l_cont_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if (pos_fi == pos_f_cut) {
                                fragArray->pos[id_frag] = 0;
                                fragArray->sub_pos[id_frag] = 0;
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = 0;
                                fragArray->len_bp[id_frag] = len_bp_f_cut;
                                fragArray->sub_len[id_frag] = sub_len_f_cut;
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = -1;
                                fragArray->next[id_frag] = id_next_f_cut;
                                fragArray->l_cont[id_frag] = l_cont_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut;
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if (pos_fi > pos_f_cut) {
                                fragArray->pos[id_frag] = pos_fi - pos_f_cut;
                                fragArray->sub_pos[id_frag] = sub_pos_fi - sub_pos_f_cut;  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = start_bp_fi - start_bp_f_cut;
                                fragArray->len_bp[id_frag] = len_bp_fi;  // UPDATE
                                fragArray->sub_len[id_frag] = sub_len_fi;
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = id_prev_fi;
                                if (id_frag == id_prev_f_cut) {
                                    fragArray->next[id_frag] = -1;
                                }
                                else {
                                    fragArray->next[id_frag] = id_next_fi;
                                }
//                                fragArray->next[id_frag] = id_next_fi;
                                fragArray->l_cont[id_frag] = l_cont_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                        }
                        else {
                            if (pos_fi < pos_f_cut) {
                                fragArray->pos[id_frag] = (l_cont_f_cut - (pos_f_cut  + 1)) + pos_fi;
                                fragArray->sub_pos[id_frag] = (sub_l_cont_f_cut - (sub_pos_f_cut  + sub_len_f_cut))
                                                              + sub_pos_fi;  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = (l_cont_bp_f_cut - (start_bp_f_cut + len_bp_f_cut))
                                                               + start_bp_fi;
                                fragArray->len_bp[id_frag] = len_bp_fi;
                                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
//                                fragArray->prev[id_frag] = id_prev_fi;
                                if (id_frag == id_next_f_cut) {
                                    fragArray->prev[id_frag] = -1;
                                }
                                else {
                                    fragArray->prev[id_frag] = id_prev_fi;
                                }
                                fragArray->next[id_frag] = id_next_fi;
                                fragArray->l_cont[id_frag] = l_cont_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if (pos_fi == pos_f_cut) {
                                fragArray->pos[id_frag] = (l_cont_f_cut - (pos_f_cut  + 1)) + pos_fi;
                                fragArray->sub_pos[id_frag] = (sub_l_cont_f_cut - (sub_pos_f_cut  + sub_len_f_cut))
                                                              + sub_pos_f_cut;  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = (l_cont_bp_f_cut - (start_bp_f_cut + len_bp_f_cut))
                                                               + start_bp_f_cut;
                                fragArray->len_bp[id_frag] = len_bp_f_cut;
                                fragArray->sub_len[id_frag] = sub_len_f_cut;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                fragArray->prev[id_frag] = id_prev_f_cut;
                                fragArray->next[id_frag] = -1;
                                fragArray->l_cont[id_frag] = l_cont_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                            else if (pos_fi > pos_f_cut) {
                                fragArray->pos[id_frag] = pos_fi - (pos_f_cut + 1);
                                fragArray->sub_pos[id_frag] = sub_pos_fi - (sub_pos_f_cut + sub_len_f_cut);  // UPDATE
                                fragArray->id_c[id_frag] = contig_f_cut;
                                split_id_contigs[id_frag] = contig_f_cut;
                                fragArray->start_bp[id_frag] = start_bp_fi - (start_bp_f_cut + len_bp_f_cut);
                                fragArray->len_bp[id_frag] = len_bp_fi;
                                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                                fragArray->circ[id_frag] = 0;
                                fragArray->id[id_frag] = id_frag;
                                fragArray->ori[id_frag] = or_fi;
                                if (pos_fi == pos_f_cut +1) {
                                    fragArray->prev[id_frag] = -1;
                                }
                                else {
                                    fragArray->prev[id_frag] = id_prev_fi;
                                }
                                fragArray->next[id_frag] = id_next_fi;
                                fragArray->l_cont[id_frag] = l_cont_f_cut;
                                fragArray->l_cont_bp[id_frag] = l_cont_bp_f_cut;
                                fragArray->sub_l_cont[id_frag] = sub_l_cont_f_cut;  // UPDATE
                                fragArray->id_d[id_frag] = id_d_fi;
                                fragArray->activ[id_frag] = activ_fi;
                                fragArray->rep[id_frag] = rep_fi;
                            }
                        }
                    }
                }
                else {
                    fragArray->pos[id_frag] = pos_fi;
                    fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                    fragArray->id_c[id_frag] = contig_fi;
                    split_id_contigs[id_frag] = contig_fi;
                    fragArray->start_bp[id_frag] = start_bp_fi;
                    fragArray->len_bp[id_frag] = len_bp_fi;
                    fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                    fragArray->circ[id_frag] = circ_fi;
                    fragArray->id[id_frag] = id_frag;
                    fragArray->ori[id_frag] = or_fi;
                    fragArray->prev[id_frag] = id_prev_fi;
                    fragArray->next[id_frag] = id_next_fi;
                    fragArray->l_cont[id_frag] = l_cont_fi;
                    fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                    fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;  // UPDATE
                    fragArray->id_d[id_frag] = id_d_fi;
                    fragArray->activ[id_frag] = activ_fi;
                    fragArray->rep[id_frag] = rep_fi;
                }
            }
            else {
                fragArray->pos[id_frag] = pos_fi;
                fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                fragArray->id_c[id_frag] = contig_fi;
                split_id_contigs[id_frag] = contig_fi;
                fragArray->start_bp[id_frag] = start_bp_fi;
                fragArray->len_bp[id_frag] = len_bp_fi;
                fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                fragArray->circ[id_frag] = circ_fi;
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = or_fi;
                fragArray->prev[id_frag] = id_prev_fi;
                fragArray->next[id_frag] = id_next_fi;
                fragArray->l_cont[id_frag] = l_cont_fi;
                fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                fragArray->sub_l_cont[id_frag] = sub_l_cont_fi;  // UPDATE
                fragArray->id_d[id_frag] = id_d_fi;
                fragArray->activ[id_frag] = activ_fi;
                fragArray->rep[id_frag] = rep_fi;
            }
        }
    }

    __global__ void paste_contigs(frag* fragArray,frag* o_fragArray, int id_fA, int id_fB, int max_id_contig,
                                  int n_frags)
    {
        __shared__ int contig_fA;
        __shared__ int pos_fA;
        __shared__ int sub_pos_fA;  // UPDATE
        __shared__ int l_cont_fA;
        __shared__ int l_cont_bp_fA;
        __shared__ int sub_l_cont_fA;  // UPDATE
        __shared__ int len_bp_fA;
        __shared__ int sub_len_fA;  // UPDATE
        __shared__ int start_bp_fA;
        __shared__ int id_prev_fA;
        __shared__ int id_next_fA;
        __shared__ int circ_fA;
        __shared__ int or_fA;
        __shared__ int activ_fA;

        __shared__ int contig_fB;
        __shared__ int pos_fB;
        __shared__ int sub_pos_fB;  // UPDATE
        __shared__ int l_cont_fB;
        __shared__ int l_cont_bp_fB;
        __shared__ int sub_l_cont_fB;  // UPDATE
        __shared__ int len_bp_fB;
        __shared__ int sub_len_fB;  // UPDATE
        __shared__ int start_bp_fB;
        __shared__ int id_prev_fB;
        __shared__ int id_next_fB;
        __shared__ int circ_fB;
        __shared__ int or_fB;
        __shared__ int activ_fB;

        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            contig_fA = o_fragArray->id_c[id_fA];
            pos_fA = o_fragArray->pos[id_fA];
            sub_pos_fA = o_fragArray->sub_pos[id_fA]; // UPDATE
            l_cont_fA = o_fragArray->l_cont[id_fA];
            l_cont_bp_fA = o_fragArray->l_cont_bp[id_fA];
            sub_l_cont_fA = o_fragArray->sub_l_cont[id_fA];  // UPDATE
            len_bp_fA = o_fragArray->len_bp[id_fA];
            sub_len_fA = o_fragArray->sub_len[id_fA];  // UPDATE
            start_bp_fA = o_fragArray->start_bp[id_fA];
            id_prev_fA = o_fragArray->prev[id_fA];
            id_next_fA = o_fragArray->next[id_fA];
            circ_fA = o_fragArray->circ[id_fA];
            or_fA = o_fragArray->ori[id_fA];
            activ_fA = o_fragArray->activ[id_fA];

            contig_fB = o_fragArray->id_c[id_fB];
            pos_fB = o_fragArray->pos[id_fB];
            sub_pos_fB = o_fragArray->sub_pos[id_fB];  // UPDATE
            l_cont_fB = o_fragArray->l_cont[id_fB];
            l_cont_bp_fB = o_fragArray->l_cont_bp[id_fB];
            sub_l_cont_fB = o_fragArray->sub_l_cont[id_fB];  // UPDATE
            len_bp_fB = o_fragArray->len_bp[id_fB];
            sub_len_fB = o_fragArray->sub_len[id_fB];  // UPDATE
            start_bp_fB = o_fragArray->start_bp[id_fB];
            id_prev_fB = o_fragArray->prev[id_fB];
            id_next_fB = o_fragArray->next[id_fB];
            circ_fB = o_fragArray->circ[id_fB];
            or_fB = o_fragArray->ori[id_fB];
            activ_fB = o_fragArray->activ[id_fB];
        }
        __syncthreads();

        int contig_fi = o_fragArray->id_c[id_frag];
        int pos_fi = o_fragArray->pos[id_frag];
        int sub_pos_fi = o_fragArray->sub_pos[id_frag];  // UPDATE
        int l_cont_fi = o_fragArray->l_cont[id_frag];
        int l_cont_bp_fi = o_fragArray->l_cont_bp[id_frag];
        int sub_l_cont_fi = o_fragArray->sub_l_cont[id_frag];  // UPDATE
        int len_bp_fi = o_fragArray->len_bp[id_frag];
        int sub_len_fi = o_fragArray->sub_len[id_frag];  // UPDATE
        int circ_fi = o_fragArray->circ[id_frag];
        int id_prev_fi = o_fragArray->prev[id_frag];
        int id_next_fi = o_fragArray->next[id_frag];
        int start_bp_fi = o_fragArray->start_bp[id_frag];
        int or_fi = o_fragArray->ori[id_frag];
        int rep_fi = o_fragArray->rep[id_frag];
        int activ_fi = o_fragArray->activ[id_frag];
        int id_d_fi = o_fragArray->id_d[id_frag];

        if (id_frag < n_frags) {
            if ( (activ_fA == 1) && ( activ_fB == 1) ) {
                if (contig_fA != contig_fB) {
                    if (contig_fi == contig_fA) {
                        if (pos_fA == 0) {
                            fragArray->pos[id_frag] = l_cont_fA - (pos_fi + 1);
                            fragArray->sub_pos[id_frag] = sub_l_cont_fA - (sub_pos_fi + sub_len_fi);  // UPDATE
                            fragArray->id_c[id_frag] = contig_fA;
                            fragArray->start_bp[id_frag] = l_cont_bp_fA - (start_bp_fi + len_bp_fi);
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi * -1;
                            if (pos_fi == l_cont_fA - 1) {
                                fragArray->prev[id_frag] = -1;
                            }
                            else {
                                fragArray->prev[id_frag] = id_next_fi;
                            }
                            if (pos_fi == pos_fA) {
                                fragArray->next[id_frag] = id_fB;
                            }
                            else {
                                fragArray->next[id_frag] = id_prev_fi;
                            }
                            fragArray->l_cont[id_frag] = l_cont_fA + l_cont_fB;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_fA + l_cont_bp_fB;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_fA + sub_l_cont_fB;  // UPDATE
                            fragArray->id_d[id_frag] = id_d_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->rep[id_frag] = rep_fi;


                        }
                        else {
                            fragArray->pos[id_frag] = pos_fi;
                            fragArray->sub_pos[id_frag] = sub_pos_fi;  // UPDATE
                            fragArray->id_c[id_frag] = contig_fA;
                            fragArray->start_bp[id_frag] = start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->prev[id_frag] = id_prev_fi;
                            if (pos_fi == pos_fA) {
                                fragArray->next[id_frag] = id_fB;
                            }
                            else {
                                fragArray->next[id_frag] = id_next_fi;
                            }
                            fragArray->l_cont[id_frag] = l_cont_fA + l_cont_fB;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_fA + l_cont_bp_fB;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_fA + sub_l_cont_fB;  // UPDATE
                            fragArray->id_d[id_frag] = id_d_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->rep[id_frag] = rep_fi;

                        }
                    }
                    else if (contig_fi == contig_fB) {
                        if (pos_fB == 0) {
                            fragArray->pos[id_frag] = l_cont_fA + pos_fi;
                            fragArray->sub_pos[id_frag] = sub_l_cont_fA + sub_pos_fi;  // UPDATE
                            fragArray->id_c[id_frag] = contig_fA;
                            fragArray->start_bp[id_frag] = l_cont_bp_fA + start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            if (pos_fi == pos_fB) {
                                fragArray->prev[id_frag] = id_fA;
                            }
                            else {
                                fragArray->prev[id_frag] = id_prev_fi;
                            }
                            fragArray->next[id_frag] = id_next_fi;
                            fragArray->l_cont[id_frag] = l_cont_fA + l_cont_fB;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_fA + l_cont_bp_fB;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_fA + sub_l_cont_fB;  // UPDATE
                            fragArray->id_d[id_frag] = id_d_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->rep[id_frag] = rep_fi;
                        }
                        else {
                            fragArray->pos[id_frag] = l_cont_fA + (l_cont_fB - (pos_fi + 1));
                            fragArray->sub_pos[id_frag] = sub_l_cont_fA + (sub_l_cont_fB - (sub_pos_fi + sub_len_fi));  // UPDATE
                            fragArray->id_c[id_frag] = contig_fA;
                            fragArray->start_bp[id_frag] = l_cont_bp_fA + (l_cont_bp_fB - (start_bp_fi + len_bp_fi));
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi;  // UPDATE
                            fragArray->circ[id_frag] = 0;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi * -1;
                            if (pos_fi == pos_fB) {
                                fragArray->prev[id_frag] = id_fA;
                            }
                            else {
                                fragArray->prev[id_frag] = id_next_fi;
                            }
                            if (pos_fi == 0) {
                                fragArray->next[id_frag] = -1;
                            }
                            else {
                                fragArray->next[id_frag] = id_prev_fi;
                            }
                            fragArray->l_cont[id_frag] = l_cont_fA + l_cont_fB;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_fA + l_cont_bp_fB;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_fA + sub_l_cont_fB; // UPDATE
                            fragArray->id_d[id_frag] = id_d_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->rep[id_frag] = rep_fi;
                        }
                    }
                    else {
                        fragArray->pos[id_frag] = pos_fi;
                        fragArray->sub_pos[id_frag] = sub_pos_fi; // UPDATE
                        fragArray->id_c[id_frag] = contig_fi;
                        fragArray->start_bp[id_frag] = start_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi; // UPDATE
                        fragArray->circ[id_frag] = circ_fi;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        fragArray->prev[id_frag] = id_prev_fi;
                        fragArray->next[id_frag] = id_next_fi;
                        fragArray->l_cont[id_frag] = l_cont_fi;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_fi; // UPDATE
                        fragArray->id_d[id_frag] = id_d_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->rep[id_frag] = rep_fi;
                    }

                }
                else if (contig_fA == contig_fB) { // circular contig
                    if (contig_fi == contig_fA) {
                        if ((pos_fA == 0) && (pos_fB == l_cont_fA - 1)) { //  creation of a circular contig !
                            fragArray->pos[id_frag] = pos_fi;
                            fragArray->sub_pos[id_frag] = sub_pos_fi; // UPDATE
                            fragArray->id_c[id_frag] = contig_fi;
                            fragArray->start_bp[id_frag] = start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi; // UPDATE
                            fragArray->circ[id_frag] = 1;
                            fragArray->ori[id_frag] = or_fi;
                            fragArray->id[id_frag] = id_frag;
                            if (pos_fi == pos_fA) {
                                fragArray->prev[id_frag] = id_fB;
                            }
                            else {
                                fragArray->prev[id_frag] = id_prev_fi;
                            }
                            if (pos_fi == l_cont_fA - 1) {
                                fragArray->next[id_frag] = id_fA;
                            }
                            else {
                                fragArray->next[id_frag] = id_next_fi;
                            }
                            fragArray->l_cont[id_frag] = l_cont_fA;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_fA;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_fA; // UPDATE
                            fragArray->id_d[id_frag] = id_d_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->rep[id_frag] = rep_fi;

                        }
                        else if((pos_fA == l_cont_fA - 1) && (pos_fB == 0)) { //  creation of a circular contig !
                            fragArray->pos[id_frag] = pos_fi;
                            fragArray->sub_pos[id_frag] = sub_pos_fi; // UPDATE
                            fragArray->id_c[id_frag] = contig_fi;
                            fragArray->start_bp[id_frag] = start_bp_fi;
                            fragArray->len_bp[id_frag] = len_bp_fi;
                            fragArray->sub_len[id_frag] = sub_len_fi; // UPDATE
                            fragArray->circ[id_frag] = 1;
                            fragArray->id[id_frag] = id_frag;
                            fragArray->ori[id_frag] = or_fi;
                            if (pos_fi == pos_fB) {
                                fragArray->prev[id_frag] = id_fA;
                            }
                            else {
                                fragArray->prev[id_frag] = id_prev_fi;
                            }
                            if (pos_fi == l_cont_fA - 1) {
                                fragArray->next[id_frag] = id_fB;
                            }
                            else {
                                fragArray->next[id_frag] = id_next_fi;
                            }
                            fragArray->l_cont[id_frag] = l_cont_fA;
                            fragArray->l_cont_bp[id_frag] = l_cont_bp_fA;
                            fragArray->sub_l_cont[id_frag] = sub_l_cont_fA; // UPDATE
                            fragArray->id_d[id_frag] = id_d_fi;
                            fragArray->activ[id_frag] = activ_fi;
                            fragArray->rep[id_frag] = rep_fi;

                        }
                    }
                    else {
                        fragArray->pos[id_frag] = pos_fi;
                        fragArray->sub_pos[id_frag] = sub_pos_fi; // UPDATE
                        fragArray->id_c[id_frag] = contig_fi;
                        fragArray->start_bp[id_frag] = start_bp_fi;
                        fragArray->len_bp[id_frag] = len_bp_fi;
                        fragArray->sub_len[id_frag] = sub_len_fi; // UPDATE
                        fragArray->circ[id_frag] = circ_fi;
                        fragArray->id[id_frag] = id_frag;
                        fragArray->ori[id_frag] = or_fi;
                        fragArray->prev[id_frag] = id_prev_fi;
                        fragArray->next[id_frag] = id_next_fi;
                        fragArray->l_cont[id_frag] = l_cont_fi;
                        fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                        fragArray->sub_l_cont[id_frag] = sub_l_cont_fi; // UPDATE
                        fragArray->id_d[id_frag] = id_d_fi;
                        fragArray->activ[id_frag] = activ_fi;
                        fragArray->rep[id_frag] = rep_fi;
                    }

                }
            }
            else {
                fragArray->pos[id_frag] = pos_fi;
                fragArray->sub_pos[id_frag] = sub_pos_fi; // UPDATE
                fragArray->id_c[id_frag] = contig_fi;
                fragArray->start_bp[id_frag] = start_bp_fi;
                fragArray->len_bp[id_frag] = len_bp_fi;
                fragArray->sub_len[id_frag] = sub_len_fi; // UPDATE
                fragArray->circ[id_frag] = circ_fi;
                fragArray->id[id_frag] = id_frag;
                fragArray->ori[id_frag] = or_fi;
                fragArray->prev[id_frag] = id_prev_fi;
                fragArray->next[id_frag] = id_next_fi;
                fragArray->l_cont[id_frag] = l_cont_fi;
                fragArray->l_cont_bp[id_frag] = l_cont_bp_fi;
                fragArray->sub_l_cont[id_frag] = sub_l_cont_fi; // UPDATE
                fragArray->id_d[id_frag] = id_d_fi;
                fragArray->activ[id_frag] = activ_fi;
                fragArray->rep[id_frag] = rep_fi;
            }
        }
    }





    __global__ void fill_vect_dist(const float4* __restrict__ subFrags2Frags,
                                   frag* fragArray,
                                   float* sub_vect_dist,
                                   int* sub_vect_id_c,
                                   float* sub_vect_s_tot,
                                   int* sub_vect_pos,
                                   int* sub_vect_len,
                                   const int* __restrict__ collector_id,
                                   const int2* __restrict__ dispatcher,
                                   const int* __restrict__ sub_collector_id,
                                   const int2* __restrict__ sub_dispatcher,
                                   int n_sub_frags,
                                   int id_mut)
    {
        int2 dispatch_fi;
        int2 sub_dispatch_fi;
        int fi, pos_i, sub_len, sub_l_cont, sub_pos_i, frag_sub_pos,  or_fi, is_activ_fi, sub_fi, is_rep_fi, swap,s_tot;
        int contig_i;
        float s, dfi, fi_start_bp;
        float4 info_fi;
        int id_rep_fi, is_circle;
        int i = 0;
        int id_pix0 = blockIdx.x * blockDim.x + threadIdx.x;
        if (id_pix0 < n_sub_frags) {
            info_fi = subFrags2Frags[id_pix0];
            dispatch_fi = dispatcher[(int) info_fi.x];
            sub_pos_i = (int) info_fi.w;
            sub_dispatch_fi = sub_dispatcher[id_pix0];
            i = 0;
            for(id_rep_fi = dispatch_fi.x; id_rep_fi < dispatch_fi.y; id_rep_fi ++) {
                fi = collector_id[id_rep_fi];
                sub_fi = sub_collector_id[sub_dispatch_fi.x + i];
                is_activ_fi = fragArray->activ[fi];
                is_rep_fi = fragArray->rep[fi];
                contig_i = fragArray->id_c[fi];
                or_fi = fragArray->ori[fi];
                pos_i = fragArray->sub_pos[fi];
                sub_len = fragArray->sub_len[fi];
                sub_l_cont = fragArray->sub_l_cont[fi];
                is_circle = fragArray->circ[fi] == 1;
                s_tot = int2float(fragArray->circ[fi]) * int2float(fragArray->l_cont_bp[fi]) / 1000.0f;
                fi_start_bp =  int2float(fragArray->start_bp[fi]) ;
//                dfi = (or_fi == 1) * info_fi.y + (or_fi != 1) * info_fi.z ;
//                df_posi = (or_fi == 1) * ( pos_i + sub_pos_i) + (or_fi != 1) * (pos_i + sub_len - sub_pos_i);
                if (or_fi == 1) {
                    dfi = info_fi.y;
                    frag_sub_pos = pos_i + sub_pos_i;
                }
                else {
                    dfi = info_fi.z;
                    frag_sub_pos = pos_i + sub_len - (sub_pos_i + 1);
                }
                sub_vect_dist[sub_fi * 12 + id_mut] = fi_start_bp / 1000.0f + dfi;
                sub_vect_id_c[sub_fi * 12 + id_mut] = contig_i;
                sub_vect_s_tot[sub_fi * 12 + id_mut] = s_tot;
                sub_vect_pos[sub_fi * 12 + id_mut] = frag_sub_pos;
                sub_vect_len[sub_fi * 12 + id_mut] = sub_l_cont;
                i += 1;
            }
        }
    }


    __global__ void uni_fill_vect_dist(const float4* __restrict__ subFrags2Frags,
                                       frag* fragArray,
                                       float* sub_vect_dist,
                                       int* sub_vect_id_c,
                                       float* sub_vect_s_tot,
                                       int* sub_vect_pos,
                                       int* sub_vect_len,
                                       const int* __restrict__ collector_id,
                                       const int2* __restrict__ dispatcher,
                                       const int* __restrict__ sub_collector_id,
                                       const int2* __restrict__ sub_dispatcher,
                                       int n_sub_frags)
    {
        int2 dispatch_fi;
        int2 sub_dispatch_fi;
        int fi, pos_i, sub_len, sub_pos_i, frag_sub_pos, or_fi, is_activ_fi, sub_fi, is_rep_fi, swap,s_tot;
        int contig_i;
        float s, dfi, fi_start_bp;
        float4 info_fi;
        int id_rep_fi, is_circle;
        int i = 0;
        int id_pix0 = blockIdx.x * blockDim.x + threadIdx.x;
        if (id_pix0 < n_sub_frags) {
            info_fi = subFrags2Frags[id_pix0];
            dispatch_fi = dispatcher[(int) info_fi.x];
            sub_pos_i = (int) info_fi.w;
            sub_dispatch_fi = sub_dispatcher[id_pix0];
            i = 0;
            for(id_rep_fi = dispatch_fi.x; id_rep_fi < dispatch_fi.y; id_rep_fi ++) {
                fi = collector_id[id_rep_fi];
                sub_fi = sub_collector_id[sub_dispatch_fi.x + i];
                is_activ_fi = fragArray->activ[fi];
                is_rep_fi = fragArray->rep[fi];
                contig_i = fragArray->id_c[fi];
                or_fi = fragArray->ori[fi];
                pos_i = fragArray->sub_pos[fi];
                sub_len = fragArray->sub_len[fi] - 1;
                is_circle = fragArray->circ[fi] == 1;
                s_tot = is_circle * int2float(fragArray->l_cont_bp[fi]) / 1000.0f;
                fi_start_bp =  int2float(fragArray->start_bp[fi]);
//                dfi = (or_fi == 1) * info_fi.y + (or_fi != 1) * info_fi.z;
                if (or_fi == 1) {
                    dfi = info_fi.y;
                    frag_sub_pos = pos_i + sub_pos_i;
                }
                else {
                    dfi = info_fi.z;
                    frag_sub_pos = pos_i + sub_len - sub_pos_i;
                }
//                frag_sub_pos = (or_fi == 1) * ( pos_i + sub_pos_i) + (or_fi != 1) * (pos_i + sub_len - sub_pos_i);
                sub_vect_dist[sub_fi] = fi_start_bp / 1000.0f + dfi;
                sub_vect_id_c[sub_fi] = contig_i;
                sub_vect_s_tot[sub_fi] = s_tot;
                sub_vect_pos[sub_fi] = frag_sub_pos;
                sub_vect_len[sub_fi] = fragArray->sub_l_cont[fi];
                i += 1;
            }
        }
    }


    __device__ inline double __shfl_down(double var, unsigned int srcLane, int width=32) {
        int2 a = *reinterpret_cast<int2*>(&var);
        a.x = __shfl_down_sync(a.x, srcLane, width);
        a.y = __shfl_down_sync(a.y, srcLane, width);
        return *reinterpret_cast<double*>(&a);
    }

    __inline__ __device__ double warpReduceSum(double val) {
        for (int offset = warpSize/2; offset > 0; offset /= 2)
            val += __shfl_down(val, offset);
        return val;
    }

    __inline__ __device__ double blockReduceSum(double val) {

        static __shared__ double shared[32]; // Shared mem for 32 partial sums
        int lane = threadIdx.x % warpSize;
        int wid = threadIdx.x / warpSize;

        val = warpReduceSum(val);     // Each warp performs partial reduction

        if (lane==0) shared[wid]=val;	// Write reduced value to shared memory

        __syncthreads();              // Wait for all partial reductions

        //read from shared memory only if that warp existed
        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0f;

        if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

        return val;
    }

    __global__ void eval_likelihood_on_zero(int* sub_vect_id_c,
                                            float* sub_vect_s_tot,
                                            int* sub_vect_pos,
                                            int* sub_vect_len,
                                            param_simu* P,
                                            float mean_size_frag,
                                            double* vect_likelihood,
                                            int* n_vals_intra,
                                            int n_frags)
    {

        __shared__ double sdata[1024]; // FERMI COMPATIBILITY
        int tid = threadIdx.x;
        int id = blockIdx.x * blockDim.x + tid;
        double val_expected = 0.0f;
        double val_likelihood = 0.0f;
        float s, s_tot, s_tot_z;
        int pos, tmp_len_cont, len_cont = 0;
        param_simu p = P[0];
        double n_tmp_vals = 0.0f;
        double tmp_likelihood = 0.0f;
        int id_frag = id;
        if (id_frag < n_frags) {
//        for (int id_frag = id; id_frag < n_frags; id_frag += blockDim.x * gridDim.x){
            pos = sub_vect_pos[id_frag];
            len_cont = sub_vect_len[id_frag];
            s_tot = sub_vect_s_tot[id_frag];
            if (pos == 0) {
                tmp_len_cont = len_cont * (len_cont - 1);
                atomicAdd(&n_vals_intra[0], tmp_len_cont / 2);
            }
            if (pos > 0) {
                s = int2float(pos) * mean_size_frag;
                s_tot_z = int2float(len_cont) * mean_size_frag;
                if (s < p.d_max) {
                    if (s_tot == 0) {
                        val_expected = (double) rippe_contacts(s, p);
                    }
                    else {
                        val_expected = (double) rippe_contacts_circ(s, s_tot_z, p);
                    }
                }
                else {
                    val_expected = (double) p.v_inter;
                }
                n_tmp_vals = __int2double_rn(len_cont -  pos);
                tmp_likelihood = val_likelihood;
                val_likelihood =  tmp_likelihood - (val_expected * n_tmp_vals);
            }
        }

//        val_likelihood = blockReduceSum(val_likelihood); // KEPLER CODE !!!
        sdata[tid] = val_likelihood;
        __syncthreads();
        for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if(threadIdx.x < offset) {
                // add a partial sum upstream to our own
                sdata[tid] += sdata[tid + offset];
            }
            // wait until all threads in the block have
            // updated their partial sums
            __syncthreads();
        }
        if (tid == 0) {
            atomicAdd(&vect_likelihood[0], sdata[0]);
        }
    }

    __global__ void eval_all_likelihood_on_zero_1st(int* sub_vect_id_c,
            float* sub_vect_s_tot,
            int* sub_vect_pos,
            int* sub_vect_len,
            param_simu* P,
            float mean_size_frag,

            int* list_uniq_mutations,
            int* n_uniq_mutations,

            double* vect_likelihood,
            int* n_vals_intra,
            int n_frags)
    {
        __shared__ double sdata[1024]; // FERMI COMPATIBILITY
        int tid = threadIdx.x;
        int id = blockIdx.x * blockDim.x + tid;
        double val_expected = 0.0f;
        double tmp_likelihood = 0.0f;
        double val_likelihood[12] = {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f};
        double tmp_val = 0.0f;
        float s, s_tot, s_tot_z;
        int id_mut, k, pos, tmp_len_cont, len_cont = 0;
        param_simu p = P[0];
        double n_tmp_vals = 0.0;
        for (int id_frag = id; id_frag < n_frags; id_frag += blockDim.x * gridDim.x) {
            for (k = 0; k < n_uniq_mutations[0]; k ++) {
                id_mut = list_uniq_mutations[k];
                pos = sub_vect_pos[id_frag * 12 + id_mut];
                len_cont = sub_vect_len[id_frag * 12 + id_mut];
                tmp_len_cont = len_cont * (len_cont -1);
                s_tot = sub_vect_s_tot[id_frag * 12 + id_mut];
                if (pos == 0) {
                    atomicAdd(&n_vals_intra[id_mut], tmp_len_cont / 2);
                }
                if (pos > 0) {
                    s = int2float(pos) * mean_size_frag;
                    s_tot_z = int2float(len_cont) * mean_size_frag;
                    if (s < p.d_max) {
                        if (s_tot == 0) {
                            val_expected = (double) rippe_contacts(s, p);
                        }
                        else {
                            val_expected = (double) rippe_contacts_circ(s, s_tot_z, p);
                        }
                    }
                    else {
                        val_expected = (double) p.v_inter;
                    }
                    n_tmp_vals = __int2double_rn(len_cont -  pos);
                    tmp_likelihood = val_likelihood[id_mut];
                    val_likelihood[id_mut] =  tmp_likelihood - val_expected * n_tmp_vals;
                }
            }
        }
//        for (id_mut=0; id_mut < 12; id_mut ++){ // KEPLER CODE !!
//            tmp_val = val_likelihood[id_mut];
//            __syncthreads();
//            tmp_val = blockReduceSum(tmp_val);
//            val_likelihood[id_mut] = tmp_val;
//        }
//        if (tid < 12){
//            atomicAdd(&vect_likelihood[tid], val_likelihood[tid]);
//        }

        for (k = 0; k < n_uniq_mutations[0]; k ++) {
            id_mut = list_uniq_mutations[k];

            sdata[tid] = val_likelihood[id_mut];
            __syncthreads();
            for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
                if(threadIdx.x < offset) {
                    // add a partial sum upstream to our own
                    sdata[tid] += sdata[tid + offset];
                }
                // wait until all threads in the block have updated their partial sums
                __syncthreads();
            }
            if (tid == 0) {
                atomicAdd(&vect_likelihood[id_mut], sdata[0]);
            }
        }
    }


    __global__ void eval_all_likelihood_on_zero_2nd(int* list_uniq_mutations,
            int* n_uniq_mutations,
            param_simu* P,
            double* vect_likelihood,
            int* n_vals_intra,
            double* n_tot_pxl)
    {
        int tid = threadIdx.x;
        int id = blockIdx.x * blockDim.x + tid;
        int id_mut;
        param_simu p = P[0];
        double intra_vals;
        double val_inter;
        double val_intra;
        double log_e =  0.43429448190325182f;
        if (tid < n_uniq_mutations[0]) {
            id_mut = list_uniq_mutations[tid];
            intra_vals = __int2double_rn(n_vals_intra[id_mut]);
            val_inter = -1.0 * log_e *  (n_tot_pxl[0] - intra_vals) * p.v_inter;
            val_intra = vect_likelihood[id_mut] * log_e;
            vect_likelihood[id_mut] = val_intra + val_inter;
        }
    }

    __global__ void eval_all_scores(int* list_uniq_mutations,
                                    int* n_uniq_mutations,
                                    double* vect_likelihood_z,
                                    double* vect_likelihood_nz,
                                    double* curr_likelihood_nz_extract,
                                    double* curr_likelihood_nz,
                                    double* vect_all_score)
    {
        int tid = threadIdx.x;
        int id = blockIdx.x * blockDim.x + tid;
        int id_mut;
        if (id < n_uniq_mutations[0]) {
            id_mut = list_uniq_mutations[tid];
            vect_all_score[id_mut] = vect_likelihood_nz[id_mut] +
                                     vect_likelihood_z[id_mut] +
                                     curr_likelihood_nz[0] - curr_likelihood_nz_extract[0];
        }
    }

    __global__ void prepare_sparse_call(const int* __restrict__ spData_row,
                                        int3* info_block,
                                        int* spData_block_csr,
                                        int *counter,
                                        int size_arr)
    {
        __shared__ int selec_smem[128];
        __shared__ int counter_smem;
        int *counter_smem_ptr;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int condition = idx < size_arr;
        int start, id_next, next, curr;
        int3 info;
        int local_count = 0;

        if ((threadIdx.x == 0))
        {
            counter_smem_ptr = &counter_smem;
            counter_smem = 0;
            info.x = 0;
            info.y = 0;
            info.z = idx;
        }
        selec_smem[threadIdx.x] = -1;
        __syncthreads();
        if (condition == 1) {
            // each counting thread writes its index to shared memory //
            id_next = min(idx + 1, size_arr - 1);
            curr = spData_row[idx];
            next = spData_row[id_next];
            if ((curr != next) || (threadIdx.x == 0) ||(threadIdx.x == 127) || (idx == size_arr - 1) ) {
                local_count = atomicAdd(counter_smem_ptr, 1);
                selec_smem[local_count] =  curr;
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            local_count = counter_smem;
            counter_smem = atomicAdd(counter, counter_smem);
            info.x = local_count;
            info.y = counter_smem;
            info.z = idx;
            info_block[blockIdx.x] = info;
        }
        __syncthreads();
        if (selec_smem[threadIdx.x] >= 0) {
            spData_block_csr[counter_smem + threadIdx.x] = selec_smem[threadIdx.x];
        }

    }

    __global__ void extract_sub_likelihood(const int* __restrict__ spData_dat,
                                           const int3* __restrict__ info_block,
                                           const int* __restrict__ spData_block_csr,
                                           const int* __restrict__ spData_row,
                                           const int* __restrict__ spData_col,
                                           param_simu* P,
                                           float mean_size_frag,
                                           float* sub_vect_pos_bp,
                                           int* sub_vect_id_c,
                                           float* sub_vect_s_tot,
                                           int* sub_vect_pos,
                                           int* sub_vect_len,

                                           double* vect_likelihood,
                                           int n_data,
                                           int n_sub_frags)
    {

        __shared__ double sdata[128]; // FERMI CODE
        __shared__ float4 all_data_row[128]; // 128
        __shared__ int list_fi[128];
        __shared__ int3 param_block;

        int tid = threadIdx.x;
        int glob_id = blockIdx.x * blockDim.x + tid;
        double loc_likelihood = 0.0f; // 128 * 12

        int curr_fi, curr_fj, fi, fj, loc_fi;
        int local_id_i = 0;
        float si, sj, s_tot, s, sf, s_z, s_tot_z, pos_i, pos_j;
        int contig_i, contig_j;
        param_simu p = P[0];
        int i;
        double dat = 0.0f;
        double val_expected = 0.0f;
        double val_expected_z = 0.0f;
        double tmp_likelihood = 0.0f;

        if (tid == 0) {
            param_block = info_block[blockIdx.x];
        }
        __syncthreads();

        int condition = (glob_id < n_data) ;

        if ((tid < param_block.x) && (condition == 1)) {
            fi = spData_block_csr[param_block.y + tid];
            list_fi[tid] = fi;
            all_data_row[tid].x = int2float(sub_vect_id_c[fi]); // contig id
            all_data_row[tid].y = sub_vect_pos_bp[fi]; // kbp position
            all_data_row[tid].z = sub_vect_s_tot[fi]; // total kb length of the contigs
            all_data_row[tid].w = int2float(sub_vect_pos[fi]); // frag position
        }

        __syncthreads();

        if ((condition == 1)) {
//            val_expected_z = 0.0f;
            curr_fi = spData_row[glob_id];
            for (i = 0; i < param_block.x; i ++) {
                if ( curr_fi == list_fi[i]) {
                    local_id_i = i;
                }
            }
            dat = (double) (spData_dat[glob_id]);
            curr_fj = spData_col[glob_id];

            fj = curr_fj;
            loc_fi = local_id_i;
            contig_i = __float2int_rd(all_data_row[loc_fi].x); // super stable ok
            contig_j = sub_vect_id_c[fj]; // ok !!


            s_tot = all_data_row[loc_fi].z;
            si = all_data_row[loc_fi].y; // stable ok
            pos_i = int2float(sub_vect_pos[curr_fi]);
            sj = sub_vect_pos_bp[fj]; // stable ...
            pos_j = int2float(sub_vect_pos[fj]);

            sf = si - sj;
            s = abs(sf);
            s_z = abs(pos_i - pos_j) * mean_size_frag;

            if (contig_i == contig_j) {
                if (s_tot == 0) {
                    val_expected = (double) rippe_contacts(s, p);
                    if (s_z < p.d_max) {
                        val_expected_z = (double) rippe_contacts(s_z, p);
                    }
                    else {
                        val_expected_z = (double) p.v_inter;
                    }
                }
                else {
                    val_expected =  (double) rippe_contacts_circ(s, s_tot, p);
                    s_tot_z = int2float(sub_vect_len[fj]) * mean_size_frag;
                    if (s_z < p.d_max) {
                        val_expected_z = (double) rippe_contacts_circ(s_z, s_tot_z, p);
                    }
                    else {
                        val_expected_z = (double) p.v_inter;
                    }
                }
            }
            else {
                val_expected = (double) p.v_inter;
                val_expected_z = (double) p.v_inter;
            }
            tmp_likelihood = evaluate_likelihood_pxl_double(val_expected, dat);
            loc_likelihood = tmp_likelihood + val_expected_z * 0.43429448190325182f;
//            loc_likelihood = tmp_likelihood;

        }
//        __syncthreads();
//        loc_likelihood = blockReduceSum(loc_likelihood); // KEPLER CODE
//        if (tid == 0){
//            atomicAdd(&vect_likelihood[0], loc_likelihood);
//        }

        sdata[tid] = loc_likelihood;
        __syncthreads();
        for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if(threadIdx.x < offset) {
                // add a partial sum upstream to our own
                sdata[tid] += sdata[tid + offset];
            }
            // wait until all threads in the block have
            // updated their partial sums
            __syncthreads();
        }

        if (tid == 0) {
            atomicAdd(&vect_likelihood[0], sdata[0]);
        }
    }


    __global__ void eval_sub_likelihood(const int* __restrict__ spData_dat,
                                        const int3* __restrict__ info_block,
                                        const int* __restrict__ spData_block_csr,
                                        const int* __restrict__ spData_row,
                                        const int* __restrict__ spData_col,
                                        param_simu* P,
                                        float mean_size_frag,
                                        const float* __restrict__ sub_vect_pos_bp,
                                        const int* __restrict__ sub_vect_id_c,
                                        const float* __restrict__ sub_vect_s_tot,
                                        const int* __restrict__ sub_vect_pos,
                                        const int* __restrict__ sub_vect_len,

                                        int* list_uniq_mutations,
                                        int* n_uniq_mutations,

                                        double* vect_likelihood,
                                        int n_data,
                                        int n_sub_frags)
    {
        __shared__ double loc_likelihood[1536]; // 128 * 12
        __shared__ float4 all_data_row[1536]    ; // 128 * 12
        __shared__ int list_fi[128];
        __shared__ int3 param_block;
        int tid = threadIdx.x;
        int tid12 = threadIdx.x * 12;
        int glob_id = blockIdx.x * blockDim.x + tid;
        int curr_fi, curr_fj, fi, fi12, fj, loc_fi;
        int local_id_i = 0;
        float si, sj, s_tot, s, sf, s_z, s_tot_z, pos_i, pos_j;
        int contig_i, contig_j;
        param_simu p = P[0];
        int i, id_mut, k;
        double dat = 0.0f;
        double val_expected = 0.0f;
        double val_expected_z = 0.0f;
        double tmp_likelihood = 0.0f;

        if (tid == 0) {
            param_block = info_block[blockIdx.x];
        }
        __syncthreads();

        int condition = (glob_id < n_data) ;

        if ((tid < param_block.x) && (condition == 1)) {
            fi = spData_block_csr[param_block.y + tid];
            fi12 = spData_block_csr[param_block.y + tid] * 12;
            list_fi[tid] = fi;
            for (i = 0; i < 12; i ++) {

                all_data_row[tid12 + i].x = int2float(sub_vect_id_c[fi12 + i]); // contig id
                all_data_row[tid12 + i].y = sub_vect_pos_bp[fi12 + i]; // kbp position
                all_data_row[tid12 + i].z = sub_vect_s_tot[fi12 + i]; // total length of the contigs
                all_data_row[tid12 + i].w = int2float(sub_vect_pos[fi12 + i]); // frag position
            }
        }

        __syncthreads();

        if ((condition == 1)) {
            curr_fi = spData_row[glob_id];
            for (i = 0; i < param_block.x; i ++) {
                if ( curr_fi == list_fi[i]) {
                    local_id_i = i * 12;
                }
            }
            dat = (double) (spData_dat[glob_id]);
            curr_fj = spData_col[glob_id];
            for (k = 0; k < n_uniq_mutations[0]; k ++) {
                id_mut = list_uniq_mutations[k];

                val_expected_z = 0.0f;
                fj = (curr_fj * 12) + id_mut;
                loc_fi = local_id_i + id_mut;
                contig_i = __float2int_rd(all_data_row[loc_fi].x); // super stable ok
                contig_j = sub_vect_id_c[fj]; // ok !!


                s_tot = all_data_row[loc_fi].z;
                si = all_data_row[loc_fi].y; // stable ok
                pos_i = all_data_row[loc_fi].w;

                sj = sub_vect_pos_bp[fj]; // stable ...
                pos_j = int2float(sub_vect_pos[fj]);

                sf = si - sj;
                s = abs(sf);
                s_z = abs(pos_i - pos_j) * mean_size_frag;


                if (contig_i == contig_j) {
                    if (s_tot == 0) {
                        val_expected = (double) rippe_contacts(s, p);
                        if (s_z < p.d_max) {
                            val_expected_z = (double) rippe_contacts(s_z, p);
                        }
                        else {
                            val_expected_z = (double) p.v_inter;
                        }
                    }
                    else {
                        val_expected =  (double) rippe_contacts_circ(s, s_tot, p);
                        if (s_z < p.d_max) {
                            s_tot_z = int2float(sub_vect_len[fj]) * mean_size_frag;
                            val_expected_z = (double) rippe_contacts_circ(s_z, s_tot_z, p);
                        }
                        else {
                            val_expected_z = (double) p.v_inter;
                        }
                    }
                }
                else {
                    val_expected = (double)  p.v_inter;
                    val_expected_z = (double) p.v_inter;
                }
                tmp_likelihood = evaluate_likelihood_pxl_double(val_expected, dat);
                loc_likelihood[tid12 + id_mut] = tmp_likelihood + val_expected_z * 0.43429448190325182f;
            }
        }
        else {
            for (id_mut = 0; id_mut < 12; id_mut ++) {
                loc_likelihood[tid12 + id_mut] =  0.0f;
            }
        }
        __syncthreads();
        if (tid < n_uniq_mutations[0]) { // tid = id mutation ok
            tmp_likelihood = 0.0f;
            id_mut = list_uniq_mutations[tid];
            for (i = 0; i < 128; i+=1) {
                tmp_likelihood += loc_likelihood[i * 12 + id_mut];
            }
            atomicAdd(&vect_likelihood[id_mut], tmp_likelihood);
        }
    }



    __global__ void evaluate_likelihood_sparse(const int* __restrict__ spData_dat,
            const int* __restrict__ spData_row,
            const int* __restrict__ spData_col,
            const int* __restrict__ id_single,

            param_simu* P,
            float mean_size_frag,

            float* sub_vect_pos_bp,
            int* sub_vect_id_c,
            float* sub_vect_s_tot,
            int* sub_vect_pos,
            int* sub_vect_len,

            double* vect_likelihood,

            int n_data_pxl,
            int n_frags)
    {
        __shared__ double sdata[1024];
        int tid = threadIdx.x;
        int id_pix0 = blockIdx.x * blockDim.x + tid;
        int fi, fj, pos_i, pos_j;
        float si, sj, s_tot, s, s_z, s_tot_z;
        int contig_i, contig_j;

        param_simu p = P[0];
        int row, col;
        int is_circle;

        double val_expected, val_expected_z, dat;
        double tmp_likelihood = 1.0;
        double loc_likelihood = 0.0f;
        int id_pix = id_pix0;

        for(id_pix = id_pix0; id_pix < n_data_pxl; id_pix += blockDim.x * gridDim.x) {

            dat = (double) (spData_dat[id_pix]);

            fi = spData_row[id_pix];
//            fi = id_single[row];
            contig_i = sub_vect_id_c[fi];
            si = sub_vect_pos_bp[fi];
            s_tot = sub_vect_s_tot[fi];
            pos_i = sub_vect_pos[fi];

            fj = spData_col[id_pix];
//            fj = id_single[col];
            contig_j = sub_vect_id_c[fj];
            sj = sub_vect_pos_bp[fj];
            pos_j = sub_vect_pos[fj];

            s = abs(si - sj);
            s_z = abs(pos_i - pos_j) * mean_size_frag;
            s_tot_z = int2float(sub_vect_len[fi]) * mean_size_frag;

            if (contig_i == contig_j) {
                if (s_tot == 0) {
                    val_expected = (double) rippe_contacts(s, p);
                    if (s_z < p.d_max) {
                        val_expected_z = (double) rippe_contacts(s_z, p);
                    }
                    else {
                        val_expected_z = (double) p.v_inter;
                    }
                }
                else {
                    val_expected =  (double) rippe_contacts_circ(s, s_tot, p);
                    if (s_z < p.d_max) {
                        val_expected_z = (double) rippe_contacts_circ(s_z, s_tot_z, p);
                    }
                    else {
                        val_expected_z = (double) p.v_inter;
                    }
                }
//                if (s_z < p.d_max){
//                    val_expected_z = (double) rippe_contacts(s_z, p);
//                }
//                else{
//                    val_expected_z = (double) p.v_inter;
//                }
            }
            else {
                val_expected = (double) p.v_inter;
                val_expected_z = (double) p.v_inter;
            }
            // log10(dat|exp) - log10(0 | exp_z)

            tmp_likelihood = evaluate_likelihood_pxl_double(val_expected, dat) + val_expected_z * 0.43429448190325182f;
//            loc_likelihood += tmp_likelihood;
            loc_likelihood += tmp_likelihood;

        }
//        __syncthreads();
//        loc_likelihood = blockReduceSum(loc_likelihood); // KEPLER CODE
//        if (tid == 0){
//            atomicAdd(&vect_likelihood[0], loc_likelihood);
//        }

        sdata[tid] = loc_likelihood;
        __syncthreads();
        for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if(threadIdx.x < offset) {
                // add a partial sum upstream to our own
                sdata[tid] += sdata[tid + offset];
            }
            // wait until all threads in the block have
            // updated their partial sums
            __syncthreads();
        }

        if (tid == 0) {
            atomicAdd(&vect_likelihood[0], sdata[0]);
        }
    }



    __global__ void extract_uniq_mutations(frag* fragArray,
                                           int frag_a,
                                           int frag_b,
                                           int* list_uniq_mutations,
                                           int* n_uniq,
                                           int flip_eject)
    {
        int id_pix = threadIdx.x + blockDim.x * blockIdx.x;
        int len_ci, len_cj;
        int n;
        int start = 0;

        if (id_pix ==0 ) {
            if (flip_eject == 1) {
                list_uniq_mutations[0] = 0;
                list_uniq_mutations[1] = 1;
                list_uniq_mutations[2] = 2;
                list_uniq_mutations[3] = 3;
                start = 4;
                n = 12;
            }
            else {
                list_uniq_mutations[0] = 2;
                list_uniq_mutations[1] = 3;
                start = 2;
                n = 10;
            }
            len_ci = fragArray->l_cont[frag_a];
            len_cj = fragArray->l_cont[frag_b];
            if (len_cj == 1) {
                n -= 4;
            }
            else {
                list_uniq_mutations[start + 0] = 4;
                list_uniq_mutations[start + 1] = 5;
                list_uniq_mutations[start + 2] = 6;
                list_uniq_mutations[start + 3] = 7;
                start += 4;
            }
            if (len_ci == 1) {
                n -= 4;
            }
            else {
                list_uniq_mutations[start + 0] = 8;
                list_uniq_mutations[start + 1] = 9;
                list_uniq_mutations[start + 2] = 10;
                list_uniq_mutations[start + 3] = 11;
            }
            n_uniq[0] = n;
        }
    }



    __global__ void set_null(float* vect, int max_id)
    {
        int id_pix = threadIdx.x + blockDim.x * blockIdx.x;
        if (id_pix < max_id) {
            vect[id_pix] = 0.0;
        }
    }


    __global__ void copy_struct(frag* fragArray, frag* smplfragArray, int* id_contigs, int n_frags)
    {
        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        int id_c = 0;
        if (id_frag  < n_frags) {
            fragArray->pos[id_frag] = smplfragArray->pos[id_frag];
            fragArray->sub_pos[id_frag] = smplfragArray->sub_pos[id_frag];
            id_c = smplfragArray->id_c[id_frag];
            fragArray->id_c[id_frag] = id_c;
            id_contigs[id_frag] = id_c;
            fragArray->circ[id_frag] = smplfragArray->circ[id_frag];
            fragArray->id[id_frag] = id_frag;
            fragArray->ori[id_frag] = smplfragArray->ori[id_frag];
            fragArray->start_bp[id_frag] = smplfragArray->start_bp[id_frag];
            fragArray->len_bp[id_frag] = smplfragArray->len_bp[id_frag];
            fragArray->sub_len[id_frag] = smplfragArray->sub_len[id_frag];
            fragArray->prev[id_frag] = smplfragArray->prev[id_frag];
            fragArray->next[id_frag] = smplfragArray->next[id_frag];
            fragArray->l_cont[id_frag] = smplfragArray->l_cont[id_frag];
            fragArray->sub_l_cont[id_frag] = smplfragArray->sub_l_cont[id_frag];
            fragArray->l_cont_bp[id_frag] = smplfragArray->l_cont_bp[id_frag];
            fragArray->rep[id_frag] = smplfragArray->rep[id_frag];
            fragArray->activ[id_frag] = smplfragArray->activ[id_frag];
            fragArray->id_d[id_frag] = smplfragArray->id_d[id_frag];
        }
    }

    __global__ void copy_gpu_array(double* dest, double* input, int max_id)
    {
        int id_pix_out = threadIdx.x + blockDim.x * blockIdx.x;
        if (id_pix_out < max_id) {
            for (int id_pix = id_pix_out; id_pix < max_id; id_pix += blockDim.x * gridDim.x) {
                dest[id_pix] = input[id_pix];
            }
        }
    }


    __global__ void simple_copy(frag* fragArray, frag* smplfragArray, int n_frags)
    {
        int id_frag = threadIdx.x + blockDim.x * blockIdx.x;
        if (id_frag  < n_frags) {
            fragArray->pos[id_frag] = smplfragArray->pos[id_frag];
            fragArray->sub_pos[id_frag] = smplfragArray->sub_pos[id_frag];
            fragArray->id_c[id_frag] = smplfragArray->id_c[id_frag];
            fragArray->circ[id_frag] = smplfragArray->circ[id_frag];
            fragArray->id[id_frag] = id_frag;
            fragArray->ori[id_frag] = smplfragArray->ori[id_frag];
            fragArray->start_bp[id_frag] = smplfragArray->start_bp[id_frag];
            fragArray->len_bp[id_frag] = smplfragArray->len_bp[id_frag];
            fragArray->sub_len[id_frag] = smplfragArray->sub_len[id_frag];
            fragArray->prev[id_frag] = smplfragArray->prev[id_frag];
            fragArray->next[id_frag] = smplfragArray->next[id_frag];
            fragArray->l_cont[id_frag] = smplfragArray->l_cont[id_frag];
            fragArray->sub_l_cont[id_frag] = smplfragArray->sub_l_cont[id_frag];
            fragArray->l_cont_bp[id_frag] = smplfragArray->l_cont_bp[id_frag];
            fragArray->rep[id_frag] = smplfragArray->rep[id_frag];
            fragArray->activ[id_frag] = smplfragArray->activ[id_frag];
            fragArray->id_d[id_frag] = smplfragArray->id_d[id_frag];
        }
    }




    __global__ void update_gpu_vect_frags(int* list_len,
                                          frag* fragArray,
                                          int * old_2_new_idx,
                                          int* id_contigs,
                                          float max_id,
                                          int n_frags,
                                          int* vect_min_id_c_new )
    {
        __shared__ float max_len;

        //get our index in the array
        int id_frag =  threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            max_len = __float2int_rd(list_len[0]);
        }
        __syncthreads();
        int min_id_c_new = vect_min_id_c_new[0];
        if (id_frag  < n_frags) {
            int id_c = fragArray->id_c[id_frag];
            int id_c_new = max_id - old_2_new_idx[id_c];
            fragArray->id_c[id_frag] = id_c_new;
            id_contigs[id_frag] = id_c_new;
        }
    }


    __global__ void gl_update_pos(int* list_len,
                                  float4* pos,
                                  float4* color,
                                  float4* vel,
                                  float4* pos_gen,
                                  float4* vel_gen,
                                  frag* fragArray,
                                  int * old_2_new_idx,
                                  int* id_contigs,
                                  float max_id,
                                  int n_frags,
                                  int id_fi,
                                  int* vect_min_id_c_new,
                                  hiprandState* state,
                                  int n_rng,
                                  float dt)
    {
        __shared__ float max_len;

        //get our index in the array
        int id_frag =  threadIdx.x + blockDim.x * blockIdx.x;
        if (threadIdx.x == 0) {
            max_len = __float2int_rd(list_len[0]);
        }
        __syncthreads();
        float min_id_c_new = int2float(vect_min_id_c_new[0]);
        if (id_frag  < n_frags) {

            int id_rng = id_frag % n_rng;
            float shift_y = (hiprand_normal(&state[id_rng]))*0.01;
            float shift_x = (hiprand_normal(&state[id_rng]))*0.01;
            float shift_z = (hiprand_normal(&state[id_rng]))*0.01;
            float shift_rot = shift_y;
            int id_c = fragArray->id_c[id_frag];
            int id_c_new = max_id - old_2_new_idx[id_c];
            fragArray->id_c[id_frag] = id_c_new;
            id_contigs[id_frag] = id_c_new;
            int is_circ = fragArray->circ[id_frag];
//            int is_circ = id_c_new % 2 == 0;
            float life = vel[id_frag].w;
            float pos_x;
            float l_cont;
            float radius;
            if (fragArray->l_cont[id_frag] > 1) {

                pos_x = (int2float(fragArray->pos[id_frag]))/ max_len;
                if (is_circ == 1) {
                    radius = (int2float(id_c_new - min_id_c_new) + shift_y * (id_frag == id_fi)) / (max_id-min_id_c_new) / 2 ;
                    l_cont = int2float(fragArray->l_cont[id_frag]) / max_len + 0.01f;
                    pos[id_frag].x = radius * 2;
                    pos[id_frag].y = 0 + radius * cos((pos_x + shift_rot) * 2 * M_PI  / l_cont ); // x plan coord
                    pos[id_frag].z = 0 + radius * sin((pos_x + shift_rot) * 2 * M_PI / l_cont ); // y plan coord;

                }
                else {
                    pos[id_frag].x = pos_x;
//                    pos[id_frag].y = ((int2float(id_c_new) - min_id_c_new) + shift_y * (id_frag == id_fi)) / (max_id-min_id_c_new);
                    pos[id_frag].y = ((int2float(id_c_new) - min_id_c_new) + shift_y * (id_frag == id_fi)) / max(1.0f,(max_id-min_id_c_new));


                    //                pos[id_frag].x = int2float(fragArray->pos[id_frag])/ max_len + 0.01f;
                    //                pos[id_frag].y = (int2float(id_c_new - min_id_c_new) + shift_y * (id_frag == id_fi)) / (max_id-min_id_c_new) + 0.01f;
                    pos[id_frag].z = 0;

                }
                color[id_frag].w = 1.5;
            }
            else {
                float4 p = pos[id_frag];
                float4 v = vel[id_frag];
                life -= dt;
                if(life <= 0.f)
                {
                    p = pos_gen[id_frag];
                    v = vel_gen[id_frag];
                    life = 1.0f;
                }
                v.z -= 9.8f*dt;

                p.x += shift_y;;
                p.y += shift_z;
                p.z += shift_x;
                v.w = life;

                //update the arrays with our newly computed values
                pos[id_frag] = p;
                vel[id_frag] = v;

                //you can manipulate the color based on properties of the system
                //here we adjust the alpha
                color[id_frag].w = life;
            }
        }
    }


    __global__ void gpu_struct_2_pxl(frag* fragArray,
                                     int* pxl_frags,
                                     int* cumul_length,
                                     int max_id,
                                     float size_im_gl,
                                     int n_frags)
    {

        int id_frag =  threadIdx.x + blockDim.x * blockIdx.x;
        float l_cont, pos, offset, tmp_pos;
        int id_c, pos_pix;

        if (id_frag < n_frags) {
            id_c = max_id - fragArray->id_c[id_frag];
            pos = int2float(fragArray->pos[id_frag]);
            l_cont = int2float(fragArray->l_cont[id_frag]);
            offset = int2float(cumul_length[id_c]);
            tmp_pos = (offset + pos) * size_im_gl / int2float(n_frags);
            pos_pix = __float2int_rz(tmp_pos);
            pxl_frags[id_frag] = pos_pix;
        }
    }


//    __global__ void fill_im_zero(unsigned char* im_gl,
////                                 hiprandState* state, int n_rng,
//                                 int size_im_gl)
//    {
//        int pix_x =  threadIdx.x + blockDim.x * blockIdx.x;
//        int pix_y =  threadIdx.y + blockDim.y * blockIdx.y;
//        if ((pix_x < pix_y) && (pix_y < size_im_gl)){
////            int id_rng = (pix_x + pix_y) % n_rng;
////            float shift_y = hiprand_normal(&state[id_rng]) * 255;
////            unsigned char out = (unsigned char) shift_y;
//            int coord = pix_x *size_im_gl + pix_y;
//            im_gl[coord] = 0;
//        }
//    }

    __global__ void update_matrix(const int* __restrict__ spData_4GL_row,
                                  const int* __restrict__ spData_4GL_col,
                                  const int* __restrict__ spData_4GL_data,
                                  const int* __restrict__ spData_block_csr,
                                  const int3* __restrict__ info_block,
                                  int* pxl_frags,
                                  int* im_gl,
                                  int size_im_gl,
                                  int n_data)
    {
        __shared__ int sdata_contact[1024];
        __shared__ int sdata_coord[1024];
        __shared__ int3 param_block;
        int tid = threadIdx.x;
        int id_pix =  tid + blockDim.x * blockIdx.x;
        int condition = id_pix < n_data;
        int x, y, local_bin, i, fi, fj, coord;

        if (tid == 0) {
            param_block = info_block[blockIdx.x];
        }

        __syncthreads();

        if ((tid < param_block.x) && (condition == 1)) {
            sdata_contact[tid] = 0;
            sdata_coord[tid] = spData_block_csr[param_block.y + tid];
        }

        __syncthreads();

        if (condition == 1) {
            fi = pxl_frags[spData_4GL_row[id_pix]];
            fj = pxl_frags[spData_4GL_col[id_pix]];
            x = min(fi, fj);
            y = max(fi, fj);
//            coord = (size_im_gl - y - 1) * size_im_gl + x;
//            coord = (y + x - 1) * size_im_gl + (x - y + size_im_gl);
            coord = x * size_im_gl + y;
            for (i = 0; i < param_block.x; i ++) {
                if ( coord == sdata_coord[i]) {
                    local_bin = i;
                }
            }
            sdata_contact[local_bin] += spData_4GL_data[id_pix];
        }
        __syncthreads();

        if ((tid < param_block.x) && (condition == 1)) {
            atomicAdd(&im_gl[sdata_coord[tid]], sdata_contact[tid]);
        }
    }



    __global__ void update_gl_buffer(unsigned char* im_gl,
                                     int* im_cuda,
                                     int thresh,
                                     int id_max)
    {
        int pix_x = threadIdx.x + blockDim.x * blockIdx.x;
        unsigned char out;
        float outf = 0.0f;
        if (pix_x < id_max) {
            outf = (float) min(im_cuda[pix_x], thresh);
            out = (unsigned char) ( outf * 255/ (float) thresh );
            im_gl[pix_x] = out;
        }
    }


    __global__ void prepare_sparse_call_4_gl(const int* __restrict__ spData_row,
            const int* __restrict__ spData_col,
            int* spData_block_csr,
            const int* __restrict__ pxl_frags,
            int3* info_block,
            int *counter,
            int size_im_gl,
            int size_arr)
    {
        __shared__ int selec_smem[1024];
        __shared__ int counter_smem;
        int *counter_smem_ptr;
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int condition = idx < size_arr;
        int condition_row, condition_col;
        int x, y, start, id_next, next_row, curr_row, next_col, curr_col, fi, next_fi, fj, next_fj, coord_pix;
        int3 info;
        int local_count = 0;

        if ((threadIdx.x == 0))
        {
            counter_smem_ptr = &counter_smem;
            counter_smem = 0;
            info.x = 0;
            info.y = 0;
            info.z = idx;
        }
        selec_smem[threadIdx.x] = -1;
        __syncthreads();
        if (condition == 1) {
            // each counting thread writes its index to shared memory //
            id_next = min(idx + 1, size_arr - 1);
            curr_row = spData_row[idx];
            next_row = spData_row[id_next];
            curr_col = spData_col[idx];
            next_col = spData_col[id_next];
            fi = pxl_frags[curr_row];
            next_fi = pxl_frags[next_row];
            fj = pxl_frags[curr_col];
            next_fj = pxl_frags[next_col];

            condition_row = fi != next_fi;
            condition_col = fj != next_fj;
//            coord_pix = min(fi, fj) * size_im_gl + max(fi, fj);
            x = min(fi, fj);
            y = max(fi, fj);
//            coord_pix = x * size_im_gl + y;
            coord_pix = (size_im_gl - y  - 1) * size_im_gl + x;
//             ret[i, j] = matrix[n - j - 1, i]; //90 degrees
//              RM[x+y+1][−x+y+n] // 45
            if ( (condition_row || condition_col) || (threadIdx.x == 0) || (threadIdx.x == 1023) || (idx == size_arr - 1) ) {
                local_count = atomicAdd(counter_smem_ptr, 1);
                selec_smem[local_count] =  coord_pix;
            }
        }
        __syncthreads();
        if (threadIdx.x == 0) {
            local_count = counter_smem;
            counter_smem = atomicAdd(counter, counter_smem);
            info.x = local_count;
            info.y = counter_smem;
            info.z = idx;
            info_block[blockIdx.x] = info;
        }
        __syncthreads();
        if (selec_smem[threadIdx.x] >= 0) {
            spData_block_csr[counter_smem + threadIdx.x] = selec_smem[threadIdx.x];
        }

    }



//    __global__ void csr_likelihood(const float* __restrict obsData2D,
//                                           frag* fragArray,
//                                           int* collector_id,
//                                           int2* dispatcher,
//                                           int4* id_sub_frags,
//                                           int4* rep_id_sub_frags,
//                                           float3* len_bp_sub_frags,
//                                           int3* accu_sub_frags,
//                                           double* likelihood,
//                                           param_simu* P,
//                                           int max_id_up_diag,
//                                           int max_id,
//                                           int n_bins,
//                                           int width_matrix,
//                                           float n_frags_per_bins)
//    {
//
//
//
//
//
//    }
} // extern "C"
